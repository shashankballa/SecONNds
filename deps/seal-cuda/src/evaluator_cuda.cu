#include "hip/hip_runtime.h"
#include "evaluator_cuda.cuh"

#include "utils/scalingvariant_cuda.cuh"

using std::invalid_argument;
using std::logic_error;

#define KERNEL_CALL(funcname, n) size_t block_count = kernel_util::ceilDiv_(n, 256); funcname<<<block_count, 256>>>
#define POLY_ARRAY_ARGUMENTS size_t poly_size, size_t coeff_modulus_size, size_t poly_modulus_degree
#define POLY_ARRAY_ARGCALL poly_size, coeff_modulus_size, poly_modulus_degree
#define GET_INDEX size_t gindex = blockDim.x * blockIdx.x + threadIdx.x
#define GET_INDEX_COND_RETURN(n) size_t gindex = blockDim.x * blockIdx.x + threadIdx.x; if (gindex >= (n)) return
#define FOR_N(name, count) for (size_t name = 0; name < count; name++)

namespace troy {

    using namespace util;

    namespace {

        inline bool areClose(double value1, double value2) {
            double scale_factor = std::max({std::fabs(value1), std::fabs(value2), 1.0});
            return std::fabs(value1 - value2) < std::numeric_limits<double>::epsilon() * scale_factor;
        }

        template <typename T, typename S>
        inline bool areSameScale(const T &value1, const S &value2) noexcept
        {
            return areClose(value1.scale(), value2.scale());
        }

        inline bool isScaleWithinBounds(
            double scale, const SEALContextCuda::ContextDataCuda &context_data) noexcept
        {
            int scale_bit_count_bound = 0;
            switch (context_data.parms().scheme())
            {
            case SchemeType::bfv:
            case SchemeType::bgv:
                scale_bit_count_bound = context_data.parms().plainModulus().bitCount();
                break;
            case SchemeType::ckks:
                scale_bit_count_bound = context_data.totalCoeffModulusBitCount();
                break;
            default:
                // Unsupported scheme; check will fail
                scale_bit_count_bound = -1;
            };

            return !(scale <= 0 || (static_cast<int>(log2(scale)) >= scale_bit_count_bound));
        }
        
        inline auto balanceCorrectionFactors(
            uint64_t factor1, uint64_t factor2, const Modulus &plain_modulus) -> std::tuple<uint64_t, uint64_t, uint64_t>
        {
            uint64_t t = plain_modulus.value();
            uint64_t half_t = t / 2;

            auto sum_abs = [&](uint64_t x, uint64_t y) {
                int64_t x_bal = static_cast<int64_t>(x > half_t ? x - t : x);
                int64_t y_bal = static_cast<int64_t>(y > half_t ? y - t : y);
                return abs(x_bal) + abs(y_bal);
            };

            // ratio = f2 / f1 mod p
            uint64_t ratio = 1;
            if (!util::tryInvertUintMod(factor1, plain_modulus, ratio))
            {
                throw std::logic_error("invalid correction factor1");
            }
            ratio = util::multiplyUintMod(ratio, factor2, plain_modulus);
            uint64_t e1 = ratio;
            uint64_t e2 = 1;
            int64_t sum = sum_abs(e1, e2);

            // Extended Euclidean
            int64_t prev_a = static_cast<int64_t>(plain_modulus.value());
            int64_t prev_b = static_cast<int64_t>(0);
            int64_t a = static_cast<int64_t>(ratio);
            int64_t b = 1;

            while (a != 0)
            {
                int64_t q = prev_a / a;
                int64_t temp = prev_a % a;
                prev_a = a;
                a = temp;

                temp = util::sub_safe(prev_b, util::mul_safe(b, q));
                prev_b = b;
                b = temp;

                uint64_t a_mod = util::barrettReduce64(static_cast<uint64_t>(abs(a)), plain_modulus);
                if (a < 0)
                {
                    a_mod = util::negateUintMod(a_mod, plain_modulus);
                }
                uint64_t b_mod = util::barrettReduce64(static_cast<uint64_t>(abs(b)), plain_modulus);
                if (b < 0)
                {
                    b_mod = util::negateUintMod(b_mod, plain_modulus);
                }
                if (a_mod != 0 && util::gcd(a_mod, t) == 1) // which also implies gcd(b_mod, t) == 1
                {
                    int64_t new_sum = sum_abs(a_mod, b_mod);
                    if (new_sum < sum)
                    {
                        sum = new_sum;
                        e1 = a_mod;
                        e2 = b_mod;
                    }
                }
            }
            return std::make_tuple(util::multiplyUintMod(e1, factor1, plain_modulus), e1, e2);
        }

        [[maybe_unused]] void printDeviceArray(const DeviceArray<uint64_t>& r, bool dont_compress = false) {
            HostArray<uint64_t> start = r.toHost();
            size_t count = r.size();
            std::cout << "dev[";
            for (size_t i = 0; i < count; i++) {
                if (!dont_compress && i == 5 && count >= 10) 
                    {i = count - 5; std::cout << "...";}
                std::cout << std::hex << start[i];
                if (i!=count-1) std::cout << ", ";
            }
            std::cout << "]\n";
        }

        [[maybe_unused]] void printDeviceArray(const uint64_t* r, size_t count, bool dont_compress = false) {
            HostArray<uint64_t> start(count);
            KernelProvider::retrieve(start.get(), r, count);
            std::cout << "dev[";
            for (size_t i = 0; i < count; i++) {
                if (!dont_compress && i == 5 && count >= 10) 
                    {i = count - 5; std::cout << "...";}
                std::cout << std::hex << start[i];
                if (i!=count-1) std::cout << ", ";
            }
            std::cout << "]\n";
        }

    }

    void EvaluatorCuda::negateInplace(CiphertextCuda& encrypted) const {
        encrypted.seed() = 0;

        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t encrypted_size = encrypted.size();

        kernel_util::kNegatePolyCoeffmod(encrypted.data(), encrypted_size, coeff_modulus.size(), parms.polyModulusDegree(), coeff_modulus, encrypted.data());
    }

    void EvaluatorCuda::addInplace(CiphertextCuda& encrypted1, const CiphertextCuda& encrypted2) const {
        encrypted1.seed() = 0;

        auto &context_data = *context_.getContextData(encrypted1.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        auto &plain_modulus = parms.plainModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted1_size = encrypted1.size();
        size_t encrypted2_size = encrypted2.size();
        size_t max_count = max(encrypted1_size, encrypted2_size);
        size_t min_count = min(encrypted1_size, encrypted2_size);

        if (encrypted1.correctionFactor() != encrypted2.correctionFactor())
        {
            // Balance correction factors and multiply by scalars before addition in BGV
            auto factors = balanceCorrectionFactors(
                encrypted1.correctionFactor(), encrypted2.correctionFactor(), plain_modulus);
            kernel_util::kMultiplyPolyScalarCoeffmod(
                encrypted1.data(), encrypted1.size(), coeff_modulus_size, coeff_count, std::get<1>(factors),
                coeff_modulus.asPointer(), encrypted1.data());

            CiphertextCuda& encrypted2_copy = temp_ciphertext_;
            encrypted2_copy = encrypted2;
            kernel_util::kMultiplyPolyScalarCoeffmod(
                encrypted2.data(), encrypted2.size(), coeff_modulus_size, coeff_count, std::get<2>(factors),
                coeff_modulus.asPointer(), encrypted2_copy.data());

            // Set new correction factor
            encrypted1.correctionFactor() = std::get<0>(factors);
            encrypted2_copy.correctionFactor() = std::get<0>(factors);

            addInplace(encrypted1, encrypted2_copy);
        }
        else
        {
            // Prepare destination
            encrypted1.resize(context_, context_data.parmsID(), max_count);
            // Add ciphertexts
            kernel_util::kAddPolyCoeffmod(encrypted1.data(), encrypted2.data(), min_count, coeff_modulus_size, coeff_count, coeff_modulus.asPointer(), encrypted1.data());

            // Copy the remainding polys of the array with larger count into encrypted1
            if (encrypted1_size < encrypted2_size)
            {
                kernel_util::kSetPolyArray(
                    encrypted2.data(min_count), encrypted2_size - encrypted1_size, coeff_count, coeff_modulus_size,
                    encrypted1.data(encrypted1_size));
            }
        }

    }

    
    void EvaluatorCuda::subInplace(CiphertextCuda& encrypted1, const CiphertextCuda& encrypted2) const {
        encrypted1.seed() = 0;
        
        auto &context_data = *context_.getContextData(encrypted1.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        auto &plain_modulus = parms.plainModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted1_size = encrypted1.size();
        size_t encrypted2_size = encrypted2.size();
        size_t max_count = max(encrypted1_size, encrypted2_size);
        size_t min_count = min(encrypted1_size, encrypted2_size);

        if (encrypted1.correctionFactor() != encrypted2.correctionFactor())
        {
            // Balance correction factors and multiply by scalars before addition in BGV
            auto factors = balanceCorrectionFactors(
                encrypted1.correctionFactor(), encrypted2.correctionFactor(), plain_modulus);
            kernel_util::kMultiplyPolyScalarCoeffmod(
                encrypted1.data(), encrypted1.size(), coeff_modulus_size, coeff_count, std::get<1>(factors),
                coeff_modulus.asPointer(), encrypted1.data());

            CiphertextCuda& encrypted2_copy = temp_ciphertext_;
            encrypted2_copy = encrypted2;
            kernel_util::kMultiplyPolyScalarCoeffmod(
                encrypted2.data(), encrypted2.size(), coeff_modulus_size, coeff_count, std::get<2>(factors),
                coeff_modulus.asPointer(), encrypted2_copy.data());

            // Set new correction factor
            encrypted1.correctionFactor() = std::get<0>(factors);
            encrypted2_copy.correctionFactor() = std::get<0>(factors);

            subInplace(encrypted1, encrypted2_copy);
        }
        else
        {
            // Prepare destination
            encrypted1.resize(context_, context_data.parmsID(), max_count);
            // Add ciphertexts
            kernel_util::kSubPolyCoeffmod(encrypted1.data(), encrypted2.data(), min_count, coeff_modulus_size, coeff_count, coeff_modulus.asPointer(), encrypted1.data());

            // Copy the remainding polys of the array with larger count into encrypted1
            if (encrypted1_size < encrypted2_size)
            {
                kernel_util::kNegatePolyCoeffmod(
                    encrypted2.data(min_count), encrypted2_size - encrypted1_size, coeff_count, coeff_modulus_size,
                    coeff_modulus, encrypted1.data(encrypted1_size));
            }
        }
    }

    void EvaluatorCuda::multiplyInplace(CiphertextCuda& encrypted1, const CiphertextCuda& encrypted2) const {
        encrypted1.seed() = 0;
        auto context_data_ptr = context_.firstContextData();
        switch (context_data_ptr->parms().scheme()) {
        case SchemeType::bfv:
            bfvMultiply(encrypted1, encrypted2);
            break;

        case SchemeType::ckks:
            ckksMultiply(encrypted1, encrypted2);
            break;

        case SchemeType::bgv:
            bgvMultiply(encrypted1, encrypted2);
            break;

        default:
            throw std::invalid_argument("unsupported scheme");
        }
    }

    void EvaluatorCuda::bfvMultiply(CiphertextCuda &encrypted1, const CiphertextCuda &encrypted2) const {
    
        if (encrypted1.isNttForm() || encrypted2.isNttForm())
            throw std::invalid_argument("encrypted1 or encrypted2 cannot be in NTT form");
    
        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted1.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t base_q_size = parms.coeffModulus().size();
        size_t encrypted1_size = encrypted1.size();
        size_t encrypted2_size = encrypted2.size();
        uint64_t plain_modulus = parms.plainModulus().value();

        auto rns_tool = context_data.rnsTool();
        size_t base_Bsk_size = rns_tool->baseBsk()->size();
        size_t base_Bsk_m_tilde_size = rns_tool->baseBskmTilde()->size();

        size_t dest_size = encrypted1_size + encrypted2_size - 1;

        auto base_q = parms.coeffModulus().get();
        auto base_Bsk = rns_tool->baseBsk()->base();
        
        auto base_q_ntt_tables = context_data.smallNTTTables();
        auto base_Bsk_ntt_tables = rns_tool->baseBskNttTables();

        auto coeff_power = util::getPowerOfTwo(coeff_count);
        
        encrypted1.resize(context_, context_data.parmsID(), dest_size);
        
        auto encrypted1_q = encrypted1_q_.ensure(encrypted1_size * coeff_count * base_q_size);
        auto encrypted1_Bsk = encrypted1_Bsk_.ensure(encrypted1_size * coeff_count * base_Bsk_size);
        
        auto encrypted2_q = encrypted2_q_.ensure(encrypted2_size * coeff_count * base_q_size);
        auto encrypted2_Bsk = encrypted2_Bsk_.ensure(encrypted2_size * coeff_count * base_Bsk_size);

        auto temp = temp_.ensure(encrypted1_size * coeff_count * base_Bsk_m_tilde_size);
        kernel_util::kSetPolyArray(encrypted1.data(), encrypted1_size, base_q_size, coeff_count, encrypted1_q.get());
        kernel_util::kNttNegacyclicHarveyLazy(encrypted1_q.get(), encrypted1_size, base_q_size, coeff_power, base_q_ntt_tables);
        for (size_t i = 0; i < encrypted1_size; i++) {
            rns_tool->fastbconvmTilde(encrypted1.data(i), temp.get() + i * coeff_count * base_Bsk_m_tilde_size);
            rns_tool->smMrq(temp.get() + i * coeff_count * base_Bsk_m_tilde_size, encrypted1_Bsk.get() + i * coeff_count * base_Bsk_size);
        }
        kernel_util::kNttNegacyclicHarveyLazy(encrypted1_Bsk.get(), encrypted1_size, base_Bsk_size, coeff_power, base_Bsk_ntt_tables);

        temp = temp_.ensure(encrypted2_size * coeff_count * base_Bsk_m_tilde_size);
        kernel_util::kSetPolyArray(encrypted2.data(), encrypted2_size, base_q_size, coeff_count, encrypted2_q.get());
        kernel_util::kNttNegacyclicHarveyLazy(encrypted2_q.get(), encrypted2_size, base_q_size, coeff_power, base_q_ntt_tables);
        for (size_t i = 0; i < encrypted2_size; i++) {
            rns_tool->fastbconvmTilde(encrypted2.data(i), temp.get() + i * coeff_count * base_Bsk_m_tilde_size);
            rns_tool->smMrq(temp.get() + i * coeff_count * base_Bsk_m_tilde_size, encrypted2_Bsk.get() + i * coeff_count * base_Bsk_size);
        }
        kernel_util::kNttNegacyclicHarveyLazy(encrypted2_Bsk.get(), encrypted2_size, base_Bsk_size, coeff_power, base_Bsk_ntt_tables);

        auto temp_dest_q = temp_dest_q_.ensure(dest_size * coeff_count * base_q_size);
        auto temp_dest_Bsk = temp_dest_Bsk_.ensure(dest_size * coeff_count * base_Bsk_size);
        KernelProvider::memsetZero(temp_dest_q.get(), dest_size * coeff_count * base_q_size);
        KernelProvider::memsetZero(temp_dest_Bsk.get(), dest_size * coeff_count * base_Bsk_size);

        for (size_t i = 0; i < dest_size; i++) {
            size_t curr_encrypted1_last = std::min<size_t>(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min<size_t>(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;

            {
                size_t d = coeff_count * base_q_size;
                auto shifted_in1_iter = encrypted1_q + curr_encrypted1_first * d;
                auto shifted_reversed_in2_iter = encrypted2_q + curr_encrypted2_first * d;
                auto shifted_out_iter = temp_dest_q + i * d;
                kernel_util::kDyadicConvolutionCoeffmod(shifted_in1_iter, shifted_reversed_in2_iter, steps, base_q_size, coeff_count,
                    base_q, shifted_out_iter);
            }
            {
                size_t d = coeff_count * base_Bsk_size;
                auto shifted_in1_iter = encrypted1_Bsk + curr_encrypted1_first * d;
                auto shifted_reversed_in2_iter = encrypted2_Bsk + curr_encrypted2_first * d;
                auto shifted_out_iter = temp_dest_Bsk + i * d;
                kernel_util::kDyadicConvolutionCoeffmod(shifted_in1_iter, shifted_reversed_in2_iter, steps, base_Bsk_size, coeff_count,
                    base_Bsk, shifted_out_iter);
            }
        }

        kernel_util::kInverseNttNegacyclicHarveyLazy(temp_dest_q.get(), dest_size, base_q_size, coeff_power, base_q_ntt_tables);
        kernel_util::kInverseNttNegacyclicHarveyLazy(temp_dest_Bsk.get(), dest_size, base_Bsk_size, coeff_power, base_Bsk_ntt_tables);

        auto temp_q_Bsk = temp_q_Bsk_.ensure(coeff_count * (base_q_size + base_Bsk_size));
        auto temp_Bsk = temp_Bsk_.ensure(coeff_count * base_Bsk_size);
        for (size_t i = 0; i < dest_size; i++) {
            kernel_util::kMultiplyPolyScalarCoeffmod(temp_dest_q + i * coeff_count * base_q_size, 
                1, base_q_size, coeff_count, 
                plain_modulus, base_q, temp_q_Bsk);
            kernel_util::kMultiplyPolyScalarCoeffmod(temp_dest_Bsk + i * coeff_count * base_Bsk_size, 
                1, base_Bsk_size, coeff_count, 
                plain_modulus, base_Bsk, temp_q_Bsk + base_q_size * coeff_count);
            rns_tool->fastFloor(temp_q_Bsk.get(), temp_Bsk.get());
            rns_tool->fastbconvSk(temp_Bsk.get(), encrypted1.data(i));
        }

    }
    
    void EvaluatorCuda::ckksMultiply(CiphertextCuda &encrypted1, const CiphertextCuda &encrypted2) const {
        
        if (!(encrypted1.isNttForm() && encrypted2.isNttForm()))
            throw std::invalid_argument("encrypted1 or encrypted2 must be in NTT form");

        
        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted1.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = parms.coeffModulus().size();
        size_t encrypted1_size = encrypted1.size();
        size_t encrypted2_size = encrypted2.size();

        auto& coeff_modulus = parms.coeffModulus();

        size_t dest_size = encrypted1_size + encrypted2_size - 1;

        encrypted1.resize(context_, context_data.parmsID(), dest_size);

        auto temp = temp_.ensure(dest_size * coeff_count * coeff_modulus_size);
        KernelProvider::memsetZero(temp.get(), dest_size * coeff_count * coeff_modulus_size);

        for (size_t i = 0; i < dest_size; i++) {
            size_t curr_encrypted1_last = std::min<size_t>(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min<size_t>(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;
            auto shifted_encrypted1_iter = encrypted1.data(curr_encrypted1_first);
            auto shifted_reversed_encrypted2_iter = encrypted2.data(curr_encrypted2_first);

            kernel_util::kDyadicConvolutionCoeffmod(
                shifted_encrypted1_iter,
                shifted_reversed_encrypted2_iter,
                steps, coeff_modulus_size, coeff_count,
                coeff_modulus,
                temp + i * coeff_count * coeff_modulus_size
            );

        }

        kernel_util::kSetPolyArray(temp, dest_size, 
            coeff_modulus_size, coeff_count, encrypted1.data());

        encrypted1.scale() *= encrypted2.scale();
        if (!isScaleWithinBounds(encrypted1.scale(), context_data))
            throw std::invalid_argument("scale out of bounds");

    }


    void EvaluatorCuda::bgvMultiply(CiphertextCuda &encrypted1, const CiphertextCuda &encrypted2) const
    {
        if (encrypted1.isNttForm() || encrypted2.isNttForm())
        {
            throw std::invalid_argument("encryped1 or encrypted2 must be not in NTT form");
        }

        auto &context_data = *context_.getContextData(encrypted1.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = parms.coeffModulus().size();
        size_t encrypted1_size = encrypted1.size();
        size_t encrypted2_size = encrypted2.size();
        auto ntt_table = context_data.smallNTTTables();

        size_t dest_size = encrypted1_size + encrypted2_size - 1;

        // Set up iterator for the base
        auto coeff_modulus = parms.coeffModulus().get();

        // Prepare destination
        encrypted1.resize(context_, context_data.parmsID(), dest_size);

        size_t coeff_power = getPowerOfTwo(coeff_count);

        // Convert c0 and c1 to ntt
        // Set up iterators for input ciphertexts
        DevicePointer<uint64_t> encrypted1_iter = encrypted1.data();
        kernel_util::kNttNegacyclicHarvey(encrypted1.data(), encrypted1_size, coeff_modulus_size, coeff_power, ntt_table);
        DevicePointer<uint64_t> encrypted2_iter;
        CiphertextCuda& encrypted2_cpy = temp_ciphertext_;
        if (&encrypted1 == &encrypted2)
        {
            encrypted2_iter = encrypted1.data();
        }
        else
        {
            encrypted2_cpy = encrypted2;
            kernel_util::kNttNegacyclicHarvey(encrypted2_cpy.data(), encrypted2_size, coeff_modulus_size, coeff_power, ntt_table);
            encrypted2_iter = encrypted2_cpy.data();
        }

        // Allocate temporary space for the result
        auto temp = temp_.ensure(dest_size * coeff_count * coeff_modulus_size);
        KernelProvider::memsetZero(temp.get(), dest_size * coeff_count * coeff_modulus_size);

        for (size_t i = 0; i < dest_size; i++) {
            size_t curr_encrypted1_last = std::min<size_t>(i, encrypted1_size - 1);
            size_t curr_encrypted2_first = std::min<size_t>(i, encrypted2_size - 1);
            size_t curr_encrypted1_first = i - curr_encrypted2_first;
            size_t steps = curr_encrypted1_last - curr_encrypted1_first + 1;

            size_t d = coeff_count * coeff_modulus_size;
            auto shifted_encrypted1_iter = encrypted1_iter + curr_encrypted1_first * d;
            auto shifted_reversed_encrypted2_iter = encrypted2_iter + curr_encrypted2_first * d;

            kernel_util::kDyadicConvolutionCoeffmod(
                shifted_encrypted1_iter, shifted_reversed_encrypted2_iter, steps,
                coeff_modulus_size, coeff_count, coeff_modulus, temp + i * d
            );
        }

        kernel_util::kSetPolyArray(temp.get(), dest_size, coeff_modulus_size, coeff_count, encrypted1.data());
        kernel_util::kInverseNttNegacyclicHarvey(encrypted1.data(), encrypted1.size(), coeff_modulus_size, coeff_power, ntt_table);
        encrypted1.correctionFactor() =
            multiplyUintMod(encrypted1.correctionFactor(), encrypted2.correctionFactor(), parms.plainModulus());
    }

    void EvaluatorCuda::squareInplace(CiphertextCuda& encrypted) const {
        encrypted.seed() = 0;
        auto context_data_ptr = context_.firstContextData();
        switch (context_data_ptr->parms().scheme())
        {
        case SchemeType::bfv:
            bfvSquare(encrypted);
            break;

        case SchemeType::ckks:
            ckksSquare(encrypted);
            break;

        case SchemeType::bgv:
            bgvSquare(encrypted);
            break;

        default:
            throw std::invalid_argument("unsupported scheme");
        }
    }

    void EvaluatorCuda::bfvSquare(CiphertextCuda &encrypted) const
    {
        if (encrypted.isNttForm())
        {
            throw std::invalid_argument("encrypted cannot be in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t base_q_size = parms.coeffModulus().size();
        size_t encrypted_size = encrypted.size();
        uint64_t plain_modulus = parms.plainModulus().value();

        auto rns_tool = context_data.rnsTool();
        size_t base_Bsk_size = rns_tool->baseBsk()->size();
        size_t base_Bsk_m_tilde_size = rns_tool->baseBskmTilde()->size();

        if (encrypted_size != 2)
        {
            bfvMultiply(encrypted, encrypted);
            return;
        }

        size_t dest_size = encrypted_size * 2 - 1;
        size_t coeff_power = util::getPowerOfTwo(coeff_count);

        auto base_q = parms.coeffModulus().get();
        auto base_Bsk = rns_tool->baseBsk()->base();

        auto base_q_ntt_tables = context_data.smallNTTTables();
        auto base_Bsk_ntt_tables = rns_tool->baseBskNttTables();

        encrypted.resize(context_, context_data.parmsID(), dest_size);

        auto encrypted_q = encrypted1_q_.ensure(encrypted_size * coeff_count * base_q_size);
        auto encrypted_Bsk = encrypted1_Bsk_.ensure(encrypted_size * coeff_count * base_Bsk_size);
        
        auto temp = temp_.ensure(coeff_count * base_Bsk_m_tilde_size);
        for (size_t i = 0; i < encrypted_size; i++) {
            kernel_util::kSetPolyArray(encrypted.data(i), 1, base_q_size, coeff_count, encrypted_q.get() + i * coeff_count * base_q_size);
            kernel_util::kNttNegacyclicHarveyLazy(encrypted_q.get() + i * coeff_count * base_q_size, 1, base_q_size, coeff_power, base_q_ntt_tables);
            rns_tool->fastbconvmTilde(encrypted.data(i), temp.get());
            rns_tool->smMrq(temp.get(), encrypted_Bsk.get() + i * coeff_count * base_Bsk_size);
            kernel_util::kNttNegacyclicHarveyLazy(encrypted_Bsk.get() + i * coeff_count * base_Bsk_size, 1, base_Bsk_size, coeff_power, base_Bsk_ntt_tables);
        }

        // printf("encrypted_q = "); printDeviceArray(encrypted_q);
        // printf("encrypted_Bsk = "); printDeviceArray(encrypted_Bsk);

        auto temp_dest_q = temp_dest_q_.ensure(dest_size * coeff_count * base_q_size);
        auto temp_dest_Bsk = temp_dest_Bsk_.ensure(dest_size * coeff_count * base_Bsk_size);
        KernelProvider::memsetZero(temp_dest_q.get(), dest_size * coeff_count * base_q_size);
        KernelProvider::memsetZero(temp_dest_Bsk.get(), dest_size * coeff_count * base_Bsk_size);

        kernel_util::kDyadicSquareCoeffmod(encrypted_q, base_q_size, coeff_count, base_q, temp_dest_q);
        kernel_util::kDyadicSquareCoeffmod(encrypted_Bsk, base_Bsk_size, coeff_count, base_Bsk, temp_dest_Bsk);

        kernel_util::kInverseNttNegacyclicHarveyLazy(temp_dest_q.get(), dest_size, base_q_size, coeff_power, base_q_ntt_tables);
        kernel_util::kInverseNttNegacyclicHarveyLazy(temp_dest_Bsk.get(), dest_size, base_Bsk_size, coeff_power, base_Bsk_ntt_tables);

        auto temp_q_Bsk = temp_q_Bsk_.ensure(coeff_count * (base_q_size + base_Bsk_size));
        auto temp_Bsk = temp_Bsk_.ensure(coeff_count * base_Bsk_size);
        for (size_t i = 0; i < dest_size; i++) {
            kernel_util::kMultiplyPolyScalarCoeffmod(temp_dest_q + i * coeff_count * base_q_size, 
                1, base_q_size, coeff_count, 
                plain_modulus, base_q, temp_q_Bsk);
            kernel_util::kMultiplyPolyScalarCoeffmod(temp_dest_Bsk + i * coeff_count * base_Bsk_size, 
                1, base_Bsk_size, coeff_count, 
                plain_modulus, base_Bsk, temp_q_Bsk + base_q_size * coeff_count);
            rns_tool->fastFloor(temp_q_Bsk.get(), temp_Bsk.get());
            rns_tool->fastbconvSk(temp_Bsk.get(), encrypted.data(i));
        }
    }

    void EvaluatorCuda::ckksSquare(CiphertextCuda &encrypted) const
    {
        if (!encrypted.isNttForm())
        {
            throw std::invalid_argument("encrypted must be in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = parms.coeffModulus().size();
        size_t encrypted_size = encrypted.size();

        // Optimization implemented currently only for size 2 ciphertexts
        if (encrypted_size != 2)
        {
            ckksMultiply(encrypted, encrypted);
            return;
        }

        // Determine destination.size()
        // Default is 3 (c_0, c_1, c_2)
        size_t dest_size = 3;

        // Set up iterator for the base
        auto& coeff_modulus = parms.coeffModulus();

        // Prepare destination
        encrypted.resize(context_, context_data.parmsID(), dest_size);

        // Set up iterators for input ciphertext
        auto encrypted_iter = encrypted.data();

        // Compute c1^2
        kernel_util::kDyadicSquareCoeffmod(encrypted.data(), coeff_modulus_size, coeff_count, coeff_modulus);

        // Set the scale
        encrypted.scale() *= encrypted.scale();
        if (!isScaleWithinBounds(encrypted.scale(), context_data))
        {
            throw std::invalid_argument("scale out of bounds");
        }
    }


    void EvaluatorCuda::bgvSquare(CiphertextCuda &encrypted) const
    {
        if (encrypted.isNttForm())
        {
            throw invalid_argument("encrypted cannot be in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = parms.coeffModulus().size();
        size_t encrypted_size = encrypted.size();
        auto ntt_table = context_data.smallNTTTables();

        // Optimization implemented currently only for size 2 ciphertexts
        if (encrypted_size != 2)
        {
            bgvMultiply(encrypted, encrypted);
            return;
        }

        // Determine destination.size()
        // Default is 3 (c_0, c_1, c_2)
        size_t dest_size = encrypted_size * 2 - 1;
        size_t coeff_power = util::getPowerOfTwo(coeff_count);

        // Set up iterator for the base
        auto coeff_modulus = parms.coeffModulus().get();

        // Prepare destination
        encrypted.resize(context_, context_data.parmsID(), dest_size);

        // Convert c0 and c1 to ntt
        kernel_util::kNttNegacyclicHarvey(encrypted.data(), encrypted_size, coeff_modulus_size, coeff_power, ntt_table);

        // Set up iterators for input ciphertext
        auto encrypted_iter = encrypted.data();

        auto temp = temp_.ensure(dest_size * coeff_count * coeff_modulus_size);
        KernelProvider::memsetZero(temp.get(), dest_size * coeff_count * coeff_modulus_size);

        kernel_util::kDyadicSquareCoeffmod(encrypted_iter.get(), coeff_modulus_size, coeff_count, coeff_modulus, temp);

        // Set the final result
        kernel_util::kSetPolyArray(temp.get(), dest_size, coeff_count, coeff_modulus_size, encrypted.data());

        // Convert the final output to Non-NTT form
        kernel_util::kInverseNttNegacyclicHarvey(encrypted.data(), dest_size, coeff_modulus_size, coeff_power, ntt_table);

        // Set the correction factor
        encrypted.correctionFactor() =
            multiplyUintMod(encrypted.correctionFactor(), encrypted.correctionFactor(), parms.plainModulus());
    }

    
    void EvaluatorCuda::relinearizeInternal(CiphertextCuda &encrypted, const RelinKeysCuda &relin_keys, std::size_t destination_size) const {
        encrypted.seed() = 0;

        // Verify parameters.
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (!context_data_ptr)
            throw invalid_argument("encrypted is not valid for encryption parameters");
        if (relin_keys.parmsID() != context_.keyParmsID())
            throw invalid_argument("relin_keys is not valid for encryption parameters");

        size_t encrypted_size = encrypted.size();

        // Verify parameters.
        if (destination_size < 2 || destination_size > encrypted_size)
            throw invalid_argument("destination_size must be at least 2 and less than or equal to current count");
        if (relin_keys.size() < sub_safe(encrypted_size, size_t(2)))
            throw invalid_argument("not enough relinearization keys");

        if (destination_size == encrypted_size)
            return;

        // Calculate number of relinearize_one_step calls needed
        size_t relins_needed = encrypted_size - destination_size;

        // Iterator pointing to the last component of encrypted
        auto encrypted_iter = encrypted.data(encrypted_size - 1);

        for (size_t i = 0; i < relins_needed; i++) {
            // std::cout << "encrypted_iter diff = " << std::dec << encrypted_iter - encrypted.data() << std::endl;
            this->switchKeyInplace(
                encrypted, encrypted_iter, static_cast<const KSwitchKeysCuda &>(relin_keys),
                RelinKeys::getIndex(encrypted_size - 1 - i));
            // std::cout << "relinearization " << i << ":";
            // printArray(encrypted.data(), encrypted.dynArray().size());
        }

        // Put the output of final relinearization into destination.
        // Prepare destination only at this point because we are resizing down
        // std::cout << "relin internal size = " << destination_size << std::endl;
        encrypted.resize(context_, context_data_ptr->parmsID(), destination_size);
        // std::cout << "relin internal size after = " << encrypted.dynArray().size() << std::endl;
    }




    void EvaluatorCuda::modSwitchScaleToNext(
        const CiphertextCuda &encrypted, CiphertextCuda &destination) const
    {
        destination.seed() = 0;
        // Assuming at this point encrypted is already validated.
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (context_data_ptr->parms().scheme() == SchemeType::bfv && encrypted.isNttForm())
        {
            throw invalid_argument("BFV encrypted cannot be in NTT form");
        }
        if (context_data_ptr->parms().scheme() == SchemeType::ckks && !encrypted.isNttForm())
        {
            throw invalid_argument("CKKS encrypted must be in NTT form");
        }
        if (context_data_ptr->parms().scheme() == SchemeType::bgv && encrypted.isNttForm())
        {
            throw invalid_argument("BGV encrypted cannot be in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_data_ptr;
        auto &next_context_data = *context_data.nextContextData();
        auto &next_parms = next_context_data.parms();
        auto rns_tool = context_data.rnsTool();

        size_t encrypted_size = encrypted.size();
        size_t coeff_count = next_parms.polyModulusDegree();
        size_t next_coeff_modulus_size = next_parms.coeffModulus().size();
        
        // size_t d = context_data.parms().coeffModulus().size() * context_

        CiphertextCuda& encrypted_copy = temp_ciphertext_;
        encrypted_copy = encrypted;

        switch (next_parms.scheme())
        {
        case SchemeType::bfv:
            for (size_t i = 0; i < encrypted_size; i++) 
                rns_tool->divideAndRoundqLastInplace(encrypted_copy.data(i));
            break;

        case SchemeType::ckks:
            for (size_t i = 0; i < encrypted_size; i++) 
                rns_tool->divideAndRoundqLastNttInplace(encrypted_copy.data(i), context_data.smallNTTTables());
            break;

        case SchemeType::bgv:
            for (size_t i = 0; i < encrypted_size; i++) 
                rns_tool->modTAndDivideqLastInplace(encrypted_copy.data(i));
            break;

        default:
            throw invalid_argument("unsupported scheme");
        }

        // Copy result to destination
        destination.resize(context_, next_context_data.parmsID(), encrypted_size);
        for (size_t i = 0; i < encrypted_size; i++) {
            kernel_util::kSetPolyArray(encrypted_copy.data(i), 1, next_coeff_modulus_size, coeff_count, destination.data(i));
        }

        // Set other attributes
        destination.isNttForm() = encrypted.isNttForm();
        if (next_parms.scheme() == SchemeType::ckks)
        {
            // Change the scale when using CKKS
            destination.scale() =
                encrypted.scale() / static_cast<double>(context_data.parms().coeffModulus().back().value());
        }
        else if (next_parms.scheme() == SchemeType::bgv)
        {
            // Change the correction factor when using BGV
            destination.correctionFactor() = multiplyUintMod(
                encrypted.correctionFactor(), rns_tool->invqLastModt(), next_parms.plainModulus());
        }
    }

    void EvaluatorCuda::modSwitchDropToNext(
        const CiphertextCuda &encrypted, CiphertextCuda &destination) const
    {
        destination.seed() = 0;
        // Assuming at this point encrypted is already validated.
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (context_data_ptr->parms().scheme() == SchemeType::ckks && !encrypted.isNttForm())
        {
            throw invalid_argument("CKKS encrypted must be in NTT form");
        }

        // Extract encryption parameters.
        auto &next_context_data = *context_data_ptr->nextContextData();
        auto &next_parms = next_context_data.parms();

        if (!isScaleWithinBounds(encrypted.scale(), next_context_data))
        {
            throw invalid_argument("scale out of bounds");
        }

        // q_1,...,q_{k-1}
        size_t next_coeff_modulus_size = next_parms.coeffModulus().size();
        size_t coeff_count = next_parms.polyModulusDegree();
        size_t encrypted_size = encrypted.size();

        // Size check
        if (!productFitsIn(encrypted_size, mul_safe(coeff_count, next_coeff_modulus_size)))
        {
            throw logic_error("invalid parameters");
        }

        if (&encrypted == &destination)
        {
            // Switching in-place so need temporary space
            auto temp = temp_.ensure(encrypted_size * coeff_count * next_coeff_modulus_size);

            // Copy data over to temp; only copy the RNS components relevant after modulus drop
            for (size_t i = 0; i < encrypted_size; i++) {
                kernel_util::kSetPolyArray(encrypted.data(i), 1, next_coeff_modulus_size, coeff_count,
                    temp + i * coeff_count * next_coeff_modulus_size);
            }

            // Resize destination before writing
            destination.resize(context_, next_context_data.parmsID(), encrypted_size);

            // Copy data to destination
            kernel_util::kSetPolyArray(temp, encrypted_size, coeff_count, next_coeff_modulus_size, destination.data());
            // TODO: avoid copying and temporary space allocation
        }
        else
        {
            // Resize destination before writing
            destination.resize(context_, next_context_data.parmsID(), encrypted_size);

            // Copy data over to destination; only copy the RNS components relevant after modulus drop
            for (size_t i = 0; i < encrypted_size; i++) {
                kernel_util::kSetPolyArray(encrypted.data(i), 1, next_coeff_modulus_size, coeff_count, 
                    destination.data(i));
            }
        }
        destination.isNttForm() = true;
        destination.scale() = encrypted.scale();
        destination.correctionFactor() = encrypted.correctionFactor();
    }

    void EvaluatorCuda::modSwitchDropToNext(PlaintextCuda &plain) const
    {
        // Assuming at this point plain is already validated.
        auto context_data_ptr = context_.getContextData(plain.parmsID());
        if (!plain.isNttForm())
        {
            throw invalid_argument("plain is not in NTT form");
        }
        if (!context_data_ptr->nextContextData())
        {
            throw invalid_argument("end of modulus switching chain reached");
        }

        // Extract encryption parameters.
        auto &next_context_data = *context_data_ptr->nextContextData();
        auto &next_parms = context_data_ptr->nextContextData()->parms();

        if (!isScaleWithinBounds(plain.scale(), next_context_data))
        {
            throw invalid_argument("scale out of bounds");
        }

        // q_1,...,q_{k-1}
        auto &next_coeff_modulus = next_parms.coeffModulus();
        size_t next_coeff_modulus_size = next_coeff_modulus.size();
        size_t coeff_count = next_parms.polyModulusDegree();

        // Compute destination size first for exception safety
        auto dest_size = mul_safe(next_coeff_modulus_size, coeff_count);

        plain.parmsID() = parmsIDZero;
        plain.resize(dest_size);
        plain.parmsID() = next_context_data.parmsID();
    }

    void EvaluatorCuda::modSwitchToNext(
        const CiphertextCuda &encrypted, CiphertextCuda &destination) const
    {
        destination.seed() = 0;

        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (context_.lastParmsID() == encrypted.parmsID())
        {
            throw invalid_argument("end of modulus switching chain reached");
        }

        switch (context_.firstContextData()->parms().scheme())
        {
        case SchemeType::bfv:
            // Modulus switching with scaling
            modSwitchScaleToNext(encrypted, destination);
            break;

        case SchemeType::ckks:
            // Modulus switching without scaling
            modSwitchDropToNext(encrypted, destination);
            break;

        case SchemeType::bgv:
            modSwitchScaleToNext(encrypted, destination);
            break;

        default:
            throw invalid_argument("unsupported scheme");
        }
    }

    void EvaluatorCuda::modSwitchToInplace(CiphertextCuda &encrypted, ParmsID parms_id) const
    {
        // Verify parameters.
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        auto targetContextData_ptr = context_.getContextData(parms_id);
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted is not valid for encryption parameters");
        }
        if (!targetContextData_ptr)
        {
            throw invalid_argument("parms_id is not valid for encryption parameters");
        }
        if (context_data_ptr->chainIndex() < targetContextData_ptr->chainIndex())
        {
            throw invalid_argument("cannot switch to higher level modulus");
        }

        while (encrypted.parmsID() != parms_id)
        {
            modSwitchToNextInplace(encrypted);
        }
    }

    void EvaluatorCuda::modSwitchToInplace(PlaintextCuda &plain, ParmsID parms_id) const
    {
        // Verify parameters.
        auto context_data_ptr = context_.getContextData(plain.parmsID());
        auto targetContextData_ptr = context_.getContextData(parms_id);
        if (!context_data_ptr)
        {
            throw invalid_argument("plain is not valid for encryption parameters");
        }
        if (!context_.getContextData(parms_id))
        {
            throw invalid_argument("parms_id is not valid for encryption parameters");
        }
        if (!plain.isNttForm())
        {
            throw invalid_argument("plain is not in NTT form");
        }
        if (context_data_ptr->chainIndex() < targetContextData_ptr->chainIndex())
        {
            throw invalid_argument("cannot switch to higher level modulus");
        }

        while (plain.parmsID() != parms_id)
        {
            modSwitchToNextInplace(plain);
        }
    }



    __global__ void gSwitchKeyInplaceUtilA(
        uint64_t* t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        const uint64_t* key_vector_j,
        size_t key_poly_coeff_size,
        const uint64_t* t_operand,
        size_t key_index,
        const Modulus* key_modulus
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        FOR_N(k, key_component_count) {
            uint64_t qword[2] {0, 0};
            const uint64_t* key_vector_j_k = key_vector_j + k * key_poly_coeff_size;
            kernel_util::dMultiplyUint64(t_operand[gindex], key_vector_j_k[key_index * coeff_count + gindex], qword);
            auto accumulator_l = t_poly_lazy + k * coeff_count * 2 + 2 * gindex;
            kernel_util::dAddUint128(qword, accumulator_l, qword);
            accumulator_l[0] = kernel_util::dBarrettReduce128(qword, key_modulus[key_index]);
            accumulator_l[1] = 0;
        }
    }

    __global__ void gSwitchKeyInplaceUtilB(
        uint64_t* t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        const uint64_t* key_vector_j,
        size_t key_poly_coeff_size,
        const uint64_t* t_operand,
        size_t key_index,
        const Modulus* key_modulus
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        FOR_N(k, key_component_count) {
            uint64_t qword[2] {0, 0};
            const uint64_t* key_vector_j_k = key_vector_j + k * key_poly_coeff_size;
            kernel_util::dMultiplyUint64(t_operand[gindex], key_vector_j_k[key_index * coeff_count + gindex], qword);
            auto accumulator_l = t_poly_lazy + k * coeff_count * 2 + 2 * gindex;
            kernel_util::dAddUint128(qword, accumulator_l, qword);
            accumulator_l[0] = qword[0];
            accumulator_l[1] = qword[1];
        }
    }

    __global__ void gSwitchKeyInplaceUtilC(
        const uint64_t* t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        uint64_t* t_poly_prod_iter
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        FOR_N(k, key_component_count) {
            const uint64_t* accumulator = t_poly_lazy + k * coeff_count * 2;
            t_poly_prod_iter[k * coeff_count * rns_modulus_size + gindex] = static_cast<uint64_t>(accumulator[gindex * 2]);
        }
    }

    __global__ void gSwitchKeyInplaceUtilD(
        const uint64_t* t_poly_lazy,
        size_t coeff_count,
        size_t key_component_count,
        size_t rns_modulus_size,
        uint64_t* t_poly_prod_iter,
        size_t key_index,
        const Modulus* key_modulus
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        FOR_N(k, key_component_count) {
            const uint64_t* accumulator = t_poly_lazy + k * coeff_count * 2;
            t_poly_prod_iter[k * coeff_count * rns_modulus_size + gindex] = 
                kernel_util::dBarrettReduce128(accumulator + gindex * 2, key_modulus[key_index]);
        }
    }

    __global__ void gSwitchKeyInplaceUtilE(
        const uint64_t* t_last,
        uint64_t* t_poly_prod_i,
        size_t coeff_count,
        const Modulus* plain_modulus,
        const Modulus* key_modulus,
        size_t decomp_modulus_size,
        size_t rns_modulus_size,
        uint64_t qk_inv_qp,
        uint64_t qk,
        const MultiplyUIntModOperand* modswitch_factors,
        uint64_t* encrypted_i
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        uint64_t k = kernel_util::dBarrettReduce64(t_last[gindex], *plain_modulus);
        k = kernel_util::dNegateUintMod(k, *plain_modulus);
        if (qk_inv_qp != 1) 
            k = kernel_util::dMultiplyScalarMod(k, qk_inv_qp, *plain_modulus);
        uint64_t delta = 0; uint64_t c_mod_qi = 0;
        FOR_N(j, decomp_modulus_size) {
            delta = kernel_util::dBarrettReduce64(k, key_modulus[j]);
            delta = kernel_util::dMultiplyScalarMod(delta, qk, key_modulus[j]);
            c_mod_qi = kernel_util::dBarrettReduce64(t_last[gindex], key_modulus[j]);
            const uint64_t Lqi = DeviceHelper::getModulusValue(key_modulus[j]) << 1;
            uint64_t& target = t_poly_prod_i[j * coeff_count + gindex];
            target = target + Lqi - (delta + c_mod_qi);
            target = kernel_util::dMultiplyUintMod(target, modswitch_factors[j], key_modulus[j]);
            encrypted_i[j * coeff_count + gindex] = kernel_util::dAddUintMod(target, encrypted_i[j * coeff_count + gindex], key_modulus[j]);
        }
    }

    __global__ void gSwitchKeyInplaceUtilF(
        uint64_t* t_last,
        size_t coeff_count,
        const Modulus* qk,
        const Modulus* key_modulus,
        uint64_t qk_half,
        size_t decomp_modulus_size,
        uint64_t* t_ntt // t_ntt should be at least coeff_count * decomp_modulus_size
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        t_last[gindex] = kernel_util::dBarrettReduce64(t_last[gindex] + qk_half, *qk);
        FOR_N(j, decomp_modulus_size) {
            const Modulus& qi = key_modulus[j];
            if (DeviceHelper::getModulusValue(*qk) > DeviceHelper::getModulusValue(qi)) {
                t_ntt[j * coeff_count + gindex] = kernel_util::dBarrettReduce64(t_last[gindex], qi);
            } else {
                t_ntt[j * coeff_count + gindex] = t_last[gindex];
            }
            uint64_t fix = DeviceHelper::getModulusValue(qi) - kernel_util::dBarrettReduce64(qk_half, key_modulus[j]);
            t_ntt[j * coeff_count + gindex] += fix;
        }
    }

    __global__ void gSwitchKeyInplaceUtilG(
        uint64_t* t_poly_prod_i,
        const uint64_t* t_ntt,
        size_t coeff_count,
        uint64_t* encrypted_i,
        bool is_ckks,
        size_t decomp_modulus_size,
        const Modulus* key_modulus,
        const MultiplyUIntModOperand* modswitch_factors
    ) {
        GET_INDEX_COND_RETURN(coeff_count);
        FOR_N(j, decomp_modulus_size) {
            uint64_t& dest = t_poly_prod_i[j*coeff_count + gindex];
            uint64_t qi = DeviceHelper::getModulusValue(key_modulus[j]);
            dest += ((is_ckks) ? (qi << 2) : (qi << 1)) - t_ntt[j * coeff_count + gindex];
            dest = kernel_util::dMultiplyUintMod(dest, modswitch_factors[j], key_modulus[j]);
            encrypted_i[j * coeff_count + gindex] = kernel_util::dAddUintMod(
                encrypted_i[j * coeff_count + gindex], dest, key_modulus[j]
            );
        }
    }

    
    void EvaluatorCuda::switchKeyInplace(
        CiphertextCuda &encrypted, ConstDevicePointer<uint64_t> target_iter, const KSwitchKeysCuda &kswitch_keys, size_t kswitch_keys_index) const
    {
        encrypted.seed() = 0;
        auto parms_id = encrypted.parmsID();
        auto &context_data = *context_.getContextData(parms_id);
        auto &parms = context_data.parms();
        auto &key_context_data = *context_.keyContextData();
        auto &key_parms = key_context_data.parms();
        auto scheme = parms.scheme();

        if (!context_.using_keyswitching())
            throw logic_error("keyswitching is not supported by the context");

        // Don't validate all of kswitch_keys but just check the parms_id.
        if (kswitch_keys.parmsID() != context_.keyParmsID())
            throw invalid_argument("parameter mismatch");

        if (kswitch_keys_index >= kswitch_keys.data().size())
            throw std::out_of_range("kswitch_keys_index");
        if (scheme == SchemeType::bfv && encrypted.isNttForm())
            throw invalid_argument("BFV encrypted cannot be in NTT form");
        if (scheme == SchemeType::ckks && !encrypted.isNttForm())
            throw invalid_argument("CKKS encrypted must be in NTT form");
        if (scheme == SchemeType::bgv && encrypted.isNttForm())
            throw invalid_argument("BGV encrypted cannot be in NTT form");

        // Extract encryption parameters.
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_power = getPowerOfTwo(coeff_count);
        size_t decomp_modulus_size = parms.coeffModulus().size();
        auto &key_modulus = key_parms.coeffModulus();
        auto key_modulus_cpu = key_modulus.toHost();
        size_t key_modulus_size = key_modulus.size();
        size_t rns_modulus_size = decomp_modulus_size + 1;
        auto key_ntt_tables = key_context_data.smallNTTTables();
        auto modswitch_factors = key_context_data.rnsTool()->invqLastModq();
        // Modulus* plain_modulus_cuda_support = KernelProvider::malloc<Modulus>(1);
        // KernelProvider::copy(plain_modulus_cuda_support, &(parms.plainModulus()), 1);
        ConstDevicePointer<Modulus> plain_modulus_cuda(parms.plainModulusCuda());

        // Prepare input
        auto &key_vector = kswitch_keys.data()[kswitch_keys_index];
        size_t key_component_count = key_vector[0].data().size();

        // Create a copy of target_iter
        auto t_target = switch_key_t_target_.ensure(coeff_count * decomp_modulus_size);
        kernel_util::kSetPolyArray(target_iter.get(), 1, decomp_modulus_size, coeff_count, t_target.get());

        // std::cout << "t_target: "; printDeviceArray(t_target);

        // In CKKS t_target is in NTT form; switch back to normal form
        if (scheme == SchemeType::ckks)
            kernel_util::kInverseNttNegacyclicHarvey(t_target, 1, decomp_modulus_size, coeff_power, key_ntt_tables);

        // Temporary result
        auto t_poly_prod = switch_key_t_poly_prod_.ensure(key_component_count * coeff_count * rns_modulus_size);
        KernelProvider::memsetZero(t_poly_prod.get(), key_component_count * coeff_count * rns_modulus_size);

        for (size_t i = 0; i < rns_modulus_size; i++) {
            // std::cout << "i = " << i << std::endl;
            size_t key_index = (i == decomp_modulus_size ? key_modulus_size - 1 : i);
            size_t lazy_reduction_summand_bound = size_t(SEAL_MULTIPLY_ACCUMULATE_USER_MOD_MAX);
            size_t lazy_reduction_counter = lazy_reduction_summand_bound;
            auto t_poly_lazy = switch_key_t_poly_lazy_.ensure(key_component_count * coeff_count * 2);
            KernelProvider::memsetZero(t_poly_lazy.get(), key_component_count * coeff_count * 2);
            // Semantic misuse of PolyIter; this is really pointing to the data for a single RNS factor
            auto t_ntt = switch_key_t_ntt_.ensure(coeff_count);
            for (size_t j = 0; j < decomp_modulus_size; j++) {
                ConstDevicePointer<uint64_t> t_operand;

                if ((scheme == SchemeType::ckks) && (i == j))
                    t_operand = target_iter + j * coeff_count;
                else
                {
                    if (key_modulus_cpu[j] <= key_modulus_cpu[key_index]) {
                        // printf("branch a - ");
                        kernel_util::kSetPolyArray(t_target.get() + j * coeff_count, 1, 1, coeff_count, t_ntt.get());
                    } else {
                        // printf("branch b - ");
                        kernel_util::kModuloPolyCoeffs(t_target.get() + j * coeff_count, 1, 1, coeff_count, key_modulus + key_index, t_ntt.get());
                    }
                    // printDeviceArray(t_ntt.get(), coeff_count);
                    kernel_util::kNttNegacyclicHarveyLazy(t_ntt.get(), 1, 1, coeff_power, key_ntt_tables + key_index);
                    t_operand = t_ntt.get();
                }
                
                // std::cout << "  j = " << j << std::endl;
                // std::cout << "  t_operand: "; printDeviceArray(t_operand.get(), coeff_count);

                size_t key_vector_poly_coeff_size = key_vector[j].data().polyCoeffSize();

                if (!lazy_reduction_counter) {
                    KERNEL_CALL(gSwitchKeyInplaceUtilA, coeff_count)(
                        t_poly_lazy.get(), coeff_count, key_component_count,
                        key_vector[j].data().data().get(), 
                        key_vector_poly_coeff_size,
                        t_operand.get(), key_index, key_modulus.get()
                    );
                } else {
                    KERNEL_CALL(gSwitchKeyInplaceUtilB, coeff_count)(
                        t_poly_lazy.get(), coeff_count, key_component_count,
                        key_vector[j].data().data().get(), 
                        key_vector_poly_coeff_size,
                        t_operand.get(), key_index, key_modulus.get()
                    );
                }

                if (!--lazy_reduction_counter)
                    lazy_reduction_counter = lazy_reduction_summand_bound;
            }

            // std::cout << "  t_poly_lazy: ";
            // printDeviceArray(t_poly_lazy);

            auto t_poly_prod_iter = t_poly_prod.get() + i * coeff_count;
            // PolyIter t_poly_prod_iter(t_poly_prod.get() + (I * coeff_count), coeff_count, rns_modulus_size);

            if (lazy_reduction_counter == lazy_reduction_summand_bound) {
                KERNEL_CALL(gSwitchKeyInplaceUtilC, coeff_count)(
                    t_poly_lazy.get(), coeff_count, key_component_count,
                    rns_modulus_size, t_poly_prod_iter
                );
            } else {
                KERNEL_CALL(gSwitchKeyInplaceUtilD, coeff_count)(
                    t_poly_lazy.get(), coeff_count, key_component_count,
                    rns_modulus_size, t_poly_prod_iter, key_index,
                    key_modulus.get()
                );
            }
        }
        // Accumulated products are now stored in t_poly_prod

        // std::cout << "t_poly_prod: ";
        // printDeviceArray(t_poly_prod, true);

        for (size_t i = 0; i < key_component_count; i++) {
            if (scheme == SchemeType::bgv)
            {
                const Modulus &plain_modulus = parms.plainModulus();
                // qk is the special prime
                uint64_t qk = key_modulus_cpu[key_modulus_size - 1].value();
                uint64_t qk_inv_qp = context_.keyContextData()->rnsTool()->invqLastModt();

                // Lazy reduction; this needs to be then reduced mod qi
                auto t_last = t_poly_prod + coeff_count * rns_modulus_size * i + decomp_modulus_size * coeff_count;
                kernel_util::kInverseNttNegacyclicHarvey(t_last, 1, 1, coeff_power, key_ntt_tables + (key_modulus_size - 1));
                kernel_util::kInverseNttNegacyclicHarvey(t_poly_prod + i * coeff_count * rns_modulus_size, 1, decomp_modulus_size, coeff_power, key_ntt_tables);

                KERNEL_CALL(gSwitchKeyInplaceUtilE, coeff_count)(
                    t_last.get(), t_poly_prod.get() + i * coeff_count * rns_modulus_size,
                    coeff_count, plain_modulus_cuda.get(), key_modulus.get(),
                    decomp_modulus_size, rns_modulus_size, qk_inv_qp, qk,
                    modswitch_factors, encrypted.data(i).get()
                );
            }
            else
            {
                // Lazy reduction; this needs to be then reduced mod qi
                auto t_last = t_poly_prod + coeff_count * rns_modulus_size * i + decomp_modulus_size * coeff_count;
                // std::cout << "t_last diff: " << coeff_count * rns_modulus_size * i + decomp_modulus_size * coeff_count << std::endl;
                auto t_ntt = switch_key_t_ntt_.ensure(decomp_modulus_size * coeff_count);
                KernelProvider::memsetZero(t_ntt.get(), decomp_modulus_size * coeff_count);
                // std::cout << "t_last: "; printArray(t_last.get(), coeff_count);
                kernel_util::kInverseNttNegacyclicHarvey(t_last, 1, 1, coeff_power, key_ntt_tables + (key_modulus_size - 1));

                KERNEL_CALL(gSwitchKeyInplaceUtilF, coeff_count)(
                    t_last.get(), coeff_count, key_modulus.get() + (key_modulus_size - 1),
                    key_modulus.get(), key_modulus_cpu[key_modulus_size - 1].value() >> 1,
                    decomp_modulus_size,
                    t_ntt.get()
                );
            
                if (scheme == SchemeType::ckks)
                    kernel_util::kNttNegacyclicHarveyLazy(t_ntt, 1, decomp_modulus_size, coeff_power, key_ntt_tables);
                else if (scheme == SchemeType::bfv)
                    kernel_util::kInverseNttNegacyclicHarveyLazy(
                        t_poly_prod + i * coeff_count * rns_modulus_size,
                        1, decomp_modulus_size, coeff_power,
                        key_ntt_tables
                    );

                // for (size_t j = 0; j < decomp_modulus_size; j++) {
                //     std::cout << "  t_ntt: " << j << " - ";
                //     printDeviceArray(t_ntt.get() + coeff_count * j, coeff_count);
                //     std::cout << "  t_ptr: " << j << " - "; printDeviceArray(t_poly_prod.get() + i * coeff_count * rns_modulus_size + j * coeff_count, coeff_count);
                // }


                gSwitchKeyInplaceUtilG<<<block_count, 256>>>(
                    t_poly_prod.get() + i * coeff_count * rns_modulus_size,
                    t_ntt.get(),
                    coeff_count, encrypted.data(i).get(), 
                    scheme==SchemeType::ckks, decomp_modulus_size, key_modulus.get(),
                    modswitch_factors
                );
            }
            // printf("enc %ld: ", i); printDeviceArray(encrypted.data(i).get(), key_component_count * coeff_count);
        }
    }

    

    void EvaluatorCuda::rescaleToNext(const CiphertextCuda &encrypted, CiphertextCuda &destination) const
    {
        destination.seed() = 0;
        if (context_.lastParmsID() == encrypted.parmsID())
        {
            throw invalid_argument("end of modulus switching chain reached");
        }

        switch (context_.firstContextData()->parms().scheme())
        {
        case SchemeType::bfv:
            /* Fall through */
        case SchemeType::bgv:
            throw invalid_argument("unsupported operation for scheme type");

        case SchemeType::ckks:
            // Modulus switching with scaling
            modSwitchScaleToNext(encrypted, destination);
            break;

        default:
            throw invalid_argument("unsupported scheme");
        }

    }


    void EvaluatorCuda::rescaleToInplace(CiphertextCuda &encrypted, ParmsID parms_id) const
    {
        encrypted.seed() = 0;

        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        auto targetContextData_ptr = context_.getContextData(parms_id);
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted is not valid for encryption parameters");
        }
        if (!targetContextData_ptr)
        {
            throw invalid_argument("parms_id is not valid for encryption parameters");
        }
        if (context_data_ptr->chainIndex() < targetContextData_ptr->chainIndex())
        {
            throw invalid_argument("cannot switch to higher level modulus");
        }

        switch (context_data_ptr->parms().scheme())
        {
        case SchemeType::bfv:
            /* Fall through */
        case SchemeType::bgv:
            throw invalid_argument("unsupported operation for scheme type");

        case SchemeType::ckks:
            while (encrypted.parmsID() != parms_id)
            {
                // Modulus switching with scaling
                modSwitchScaleToNext(encrypted, encrypted);
            }
            break;

        default:
            throw invalid_argument("unsupported scheme");
        }
    }


    void EvaluatorCuda::multiplyMany(
        const std::vector<CiphertextCuda> &encrypteds, const RelinKeysCuda &relin_keys, CiphertextCuda &destination) const
    {
        destination.seed() = 0;
        // Verify parameters.
        if (encrypteds.size() == 0)
        {
            throw invalid_argument("encrypteds vector must not be empty");
        }
        for (size_t i = 0; i < encrypteds.size(); i++)
        {
            if (&encrypteds[i] == &destination)
            {
                throw invalid_argument("encrypteds must be different from destination");
            }
        }

        // There is at least one ciphertext
        auto context_data_ptr = context_.getContextData(encrypteds[0].parmsID());
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypteds is not valid for encryption parameters");
        }

        // Extract encryption parameters.
        auto &context_data = *context_data_ptr;
        auto &parms = context_data.parms();

        if (parms.scheme() != SchemeType::bfv && parms.scheme() != SchemeType::bgv)
        {
            throw logic_error("unsupported scheme");
        }

        // If there is only one ciphertext, return it.
        if (encrypteds.size() == 1)
        {
            destination = encrypteds[0];
            return;
        }

        // Do first level of multiplications
        std::vector<CiphertextCuda> product_vec;
        for (size_t i = 0; i < encrypteds.size() - 1; i += 2)
        {
            CiphertextCuda temp(context_, context_data.parmsID());
            if (encrypteds[i].data().get() == encrypteds[i + 1].data().get())
            {
                // printf("square %lld %lld\n", encrypteds[i].data().get(), encrypteds[i + 1].data().get());
                square(encrypteds[i], temp);
            }
            else
            {
                // printf("mul\n");
                multiply(encrypteds[i], encrypteds[i + 1], temp);
            }
            // printf("relin\n");
            relinearizeInplace(temp, relin_keys);
            product_vec.emplace_back(std::move(temp));
            // printf("emp_back %ld\n", product_vec.size());
        }
        if (encrypteds.size() & 1)
        {
            product_vec.emplace_back(encrypteds.back());
        }

        // Repeatedly multiply and add to the back of the vector until the end is reached
        for (size_t i = 0; i < product_vec.size() - 1; i += 2)
        {
            CiphertextCuda temp(context_, context_data.parmsID());
            multiply(product_vec[i], product_vec[i + 1], temp);
            relinearizeInplace(temp, relin_keys);
            product_vec.emplace_back(std::move(temp));
            // printf("emp_back %ld\n", product_vec.size());
        }

        destination = product_vec.back();
    }

    void EvaluatorCuda::exponentiateInplace(
        CiphertextCuda &encrypted, uint64_t exponent, const RelinKeysCuda &relin_keys) const
    {
        encrypted.seed() = 0;
        // Verify parameters.
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted is not valid for encryption parameters");
        }
        if (!context_.getContextData(relin_keys.parmsID()))
        {
            throw invalid_argument("relin_keys is not valid for encryption parameters");
        }
        if (exponent == 0)
        {
            throw invalid_argument("exponent cannot be 0");
        }

        // Fast case
        if (exponent == 1)
        {
            return;
        }

        // Create a vector of copies of encrypted
        std::vector<CiphertextCuda> exp_vector(static_cast<size_t>(exponent), encrypted);
        multiplyMany(exp_vector, relin_keys, encrypted);
    }



    void EvaluatorCuda::addPlainInplace(CiphertextCuda &encrypted, const PlaintextCuda &plain) const
    {
        encrypted.seed() = 0;

        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        if (parms.scheme() == SchemeType::bfv && encrypted.isNttForm())
        {
            throw invalid_argument("BFV encrypted cannot be in NTT form");
        }
        if (parms.scheme() == SchemeType::ckks && !encrypted.isNttForm())
        {
            throw invalid_argument("CKKS encrypted must be in NTT form");
        }
        if (parms.scheme() == SchemeType::bgv && encrypted.isNttForm())
        {
            throw invalid_argument("BGV encrypted cannot be in NTT form");
        }
        if (plain.isNttForm() != encrypted.isNttForm())
        {
            throw invalid_argument("NTT form mismatch");
        }
        if (encrypted.isNttForm() && (encrypted.parmsID() != plain.parmsID()))
        {
            throw invalid_argument("encrypted and plain parameter mismatch");
        }
        if (!areSameScale(encrypted, plain))
        {
            throw invalid_argument("scale mismatch");
        }

        // Extract encryption parameters.
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        switch (parms.scheme())
        {
        case SchemeType::bfv:
        {
            multiplyAddPlainWithScalingVariant(plain, context_data, encrypted.data(0));
            break;
        }

        case SchemeType::ckks:
        {
            DevicePointer encrypted_iter(encrypted.data());
            ConstDevicePointer plain_iter(plain.data());
            kernel_util::kAddPolyCoeffmod(encrypted_iter, plain_iter, 1, coeff_modulus_size, coeff_count, coeff_modulus, encrypted_iter);
            break;
        }

        case SchemeType::bgv:
        {
            PlaintextCuda plain_copy = plain;
            kernel_util::kMultiplyPolyScalarCoeffmod(plain.data(), 
                1, 1, plain.coeffCount(), encrypted.correctionFactor(),
                parms.plainModulusCuda(), plain_copy.data());
            addPlainWithoutScalingVariant(plain_copy, context_data, encrypted.data(0));
            break;
        }

        default:
            throw invalid_argument("unsupported scheme");
        }
    }

    void EvaluatorCuda::subPlainInplace(CiphertextCuda &encrypted, const PlaintextCuda &plain) const
    {        
        encrypted.seed() = 0;

        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        if (parms.scheme() == SchemeType::bfv && encrypted.isNttForm())
        {
            throw invalid_argument("BFV encrypted cannot be in NTT form");
        }
        if (parms.scheme() == SchemeType::ckks && !encrypted.isNttForm())
        {
            throw invalid_argument("CKKS encrypted must be in NTT form");
        }
        if (parms.scheme() == SchemeType::bgv && encrypted.isNttForm())
        {
            throw invalid_argument("BGV encrypted cannot be in NTT form");
        }
        if (plain.isNttForm() != encrypted.isNttForm())
        {
            throw invalid_argument("NTT form mismatch");
        }
        if (encrypted.isNttForm() && (encrypted.parmsID() != plain.parmsID()))
        {
            throw invalid_argument("encrypted and plain parameter mismatch");
        }
        if (!areSameScale(encrypted, plain))
        {
            throw invalid_argument("scale mismatch");
        }

        // Extract encryption parameters.
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        switch (parms.scheme())
        {
        case SchemeType::bfv:
        {
            multiplySubPlainWithScalingVariant(plain, context_data, encrypted.data(0));
            break;
        }

        case SchemeType::ckks:
        {
            DevicePointer encrypted_iter(encrypted.data());
            ConstDevicePointer plain_iter(plain.data());
            kernel_util::kSubPolyCoeffmod(encrypted_iter, plain_iter, 1, coeff_modulus_size, coeff_count, coeff_modulus, encrypted_iter);
            break;
        }

        case SchemeType::bgv:
        {
            PlaintextCuda plain_copy = plain;
            kernel_util::kMultiplyPolyScalarCoeffmod(plain.data(), 
                1, 1, plain.coeffCount(), encrypted.correctionFactor(),
                parms.plainModulusCuda(), plain_copy.data());
            subPlainWithoutScalingVariant(plain_copy, context_data, encrypted.data(0));
            break;
        }

        default:
            throw invalid_argument("unsupported scheme");
        }
    }

    
    void EvaluatorCuda::multiplyPlainInplace(CiphertextCuda &encrypted, const PlaintextCuda &plain) const
    {
        encrypted.seed() = 0;
        if (encrypted.isNttForm() != plain.isNttForm())
        {
            throw invalid_argument("NTT form mismatch");
        }

        if (encrypted.isNttForm())
        {
            multiplyPlainNtt(encrypted, plain);
        }
        else
        {
            multiplyPlainNormal(encrypted, plain);
        }
    }

    __global__ void gMultiplyPlainNormalUtilA(
        const uint64_t* plain_data,
        size_t plain_coeff_count,
        const uint64_t* plain_upper_half_increment,
        uint64_t plain_upper_half_threshold,
        size_t coeff_modulus_size,
        uint64_t* temp
    ) {
        GET_INDEX_COND_RETURN(plain_coeff_count);
        uint64_t plain_value = plain_data[gindex];
        if (plain_value >= plain_upper_half_threshold) {
            kernel_util::dAddUint(plain_upper_half_increment, coeff_modulus_size, plain_value, temp + coeff_modulus_size * gindex);
        } else {
            temp[coeff_modulus_size * gindex] = plain_value;
        }
    }

    __global__ void gMultiplyPlainNormalUtilB(
        const uint64_t* plain_data,
        size_t plain_coeff_count,
        size_t coeff_count,
        const uint64_t* plain_upper_half_increment,
        uint64_t plain_upper_half_threshold,
        size_t coeff_modulus_size,
        uint64_t* temp
    ) {
        GET_INDEX_COND_RETURN(plain_coeff_count);
        uint64_t plain_value = plain_data[gindex];
        FOR_N(i, coeff_modulus_size) {
            temp[i * coeff_count + gindex] = 
                plain_value >= plain_upper_half_threshold ?
                (plain_value + plain_upper_half_increment[i]) : (plain_value);
        }
    }

    void EvaluatorCuda::multiplyPlainNormal(CiphertextCuda &encrypted, const PlaintextCuda &plain) const
    {
        encrypted.seed() = 0;
        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_power = getPowerOfTwo(coeff_count);
        size_t coeff_modulus_size = coeff_modulus.size();

        uint64_t plain_upper_half_threshold = context_data.plainUpperHalfThreshold();
        auto plain_upper_half_increment = context_data.plainUpperHalfIncrement();
        auto ntt_tables = context_data.smallNTTTables();

        size_t encrypted_size = encrypted.size();
        size_t plain_coeff_count = plain.coeffCount();

        // Size check
        if (!productFitsIn(encrypted_size, mul_safe(coeff_count, coeff_modulus_size)))
        {
            throw logic_error("invalid parameters");
        }

        // Generic case: any plaintext polynomial
        // Allocate temporary space for an entire RNS polynomial
        auto temp = temp_.ensure(coeff_count * coeff_modulus_size);
        KernelProvider::memsetZero(temp.get(), coeff_count * coeff_modulus_size);

        if (!context_data.qualifiers().using_fast_plain_lift) {
            KERNEL_CALL(gMultiplyPlainNormalUtilA, plain_coeff_count)(
                plain.data(), plain_coeff_count, plain_upper_half_increment.get(),
                plain_upper_half_threshold, coeff_modulus_size, temp.get()
            );
            context_data.rnsTool()->baseq()->decomposeArray(temp.get(), coeff_count);
        }
        else
        {
            KERNEL_CALL(gMultiplyPlainNormalUtilB, plain_coeff_count)(
                plain.data(), plain_coeff_count, coeff_count, plain_upper_half_increment.get(),
                plain_upper_half_threshold, coeff_modulus_size, temp.get()
            );
        }

        // Need to multiply each component in encrypted with temp; first step is to transform to NTT form
        // RNSIter temp_iter(temp.get(), coeff_count);
        kernel_util::kNttNegacyclicHarvey(temp.get(), 1, coeff_modulus_size, coeff_power, ntt_tables);

        for (size_t i = 0; i < encrypted_size; i++) {
            auto target_ptr = encrypted.data(i);
            kernel_util::kNttNegacyclicHarveyLazy(target_ptr, 1, coeff_modulus_size, coeff_power, ntt_tables);
            kernel_util::kDyadicProductCoeffmod(target_ptr, temp, 1, coeff_modulus_size, coeff_count, coeff_modulus, target_ptr);
            kernel_util::kInverseNttNegacyclicHarveyLazy(target_ptr, 1, coeff_modulus_size, coeff_power, ntt_tables);
        }

        // Set the scale
        if (parms.scheme() == SchemeType::ckks) {
            encrypted.scale() *= plain.scale();
            if (!isScaleWithinBounds(encrypted.scale(), context_data))
            {
                throw invalid_argument("scale out of bounds");
            }
        }
    }

    

    void EvaluatorCuda::multiplyPlainNtt(CiphertextCuda &encrypted_ntt, const PlaintextCuda &plain_ntt) const
    {
        encrypted_ntt.seed() = 0;
        // Verify parameters.
        if (!plain_ntt.isNttForm())
        {
            throw invalid_argument("plain_ntt is not in NTT form");
        }
        if (encrypted_ntt.parmsID() != plain_ntt.parmsID())
        {
            throw invalid_argument("encrypted_ntt and plain_ntt parameter mismatch");
        }

        // Extract encryption parameters.
        auto &context_data = *context_.getContextData(encrypted_ntt.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_ntt_size = encrypted_ntt.size();

        // Size check
        if (!productFitsIn(encrypted_ntt_size, mul_safe(coeff_count, coeff_modulus_size))) {
            throw logic_error("invalid parameters");
        }

        auto plain_ntt_iter = plain_ntt.data();

        for (size_t i = 0; i < encrypted_ntt_size; i++) {
            kernel_util::kDyadicProductCoeffmod(encrypted_ntt.data(i), plain_ntt_iter,
                1, coeff_modulus_size, coeff_count, coeff_modulus, encrypted_ntt.data(i));
        }

        // Set the scale
        encrypted_ntt.scale() *= plain_ntt.scale();
        if (!isScaleWithinBounds(encrypted_ntt.scale(), context_data))
        {
            throw invalid_argument("scale out of bounds");
        }
    }

    __global__ void gTransformToNttInplace(
        uint64_t* plain,
        size_t plain_coeff_count,
        size_t coeff_count,
        const uint64_t* plain_upper_half_increment,
        uint64_t plain_upper_half_threshold,
        size_t coeff_modulus_size
    ) {
        GET_INDEX_COND_RETURN(plain_coeff_count);
        FOR_N(i, coeff_modulus_size) {
            size_t plain_index = (coeff_modulus_size - 1 - i) * coeff_count + gindex;
            size_t increment_index = (coeff_modulus_size - 1 - i);
            plain[plain_index] = (plain[gindex] >= plain_upper_half_threshold) ?
                (plain[gindex] + plain_upper_half_increment[increment_index]) : plain[gindex];
        }
    }

    void EvaluatorCuda::transformToNttInplace(PlaintextCuda &plain, ParmsID parms_id) const
    {

        auto context_data_ptr = context_.getContextData(parms_id);
        if (!context_data_ptr)
        {
            throw invalid_argument("parms_id is not valid for the current context");
        }
        if (plain.isNttForm())
        {
            throw invalid_argument("plain is already in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_data_ptr;
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t plain_coeff_count = plain.coeffCount();

        uint64_t plain_upper_half_threshold = context_data.plainUpperHalfThreshold();
        auto plain_upper_half_increment = context_data.plainUpperHalfIncrement();

        auto ntt_tables = context_data.smallNTTTables();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        // Resize to fit the entire NTT transformed (ciphertext size) polynomial
        // Note that the new coefficients are automatically set to 0
        plain.resize(coeff_count * coeff_modulus_size);
        auto plain_iter = plain.data();

        if (!context_data.qualifiers().using_fast_plain_lift)
        {
            // Allocate temporary space for an entire RNS polynomial
            // Slight semantic misuse of RNSIter here, but this works well
            auto temp = temp_.ensure(coeff_modulus_size * coeff_count);
            KernelProvider::memsetZero(temp.get(), coeff_modulus_size * coeff_count);
            
            KERNEL_CALL(gMultiplyPlainNormalUtilA, plain_coeff_count)(
                plain.data(), plain_coeff_count, plain_upper_half_increment.get(),
                plain_upper_half_threshold, coeff_modulus_size, temp.get()
            );

            context_data.rnsTool()->baseq()->decomposeArray(temp.get(), coeff_count);

            // Copy data back to plain
            kernel_util::kSetPolyArray(temp.get(), 1, coeff_count, coeff_modulus_size, plain.data());
        }
        else
        {
            KERNEL_CALL(gTransformToNttInplace, plain_coeff_count)(
                plain.data(), plain_coeff_count, coeff_count, plain_upper_half_increment.get(),
                plain_upper_half_threshold, coeff_modulus_size
            );
        }

        // Transform to NTT domain
        kernel_util::kNttNegacyclicHarvey(plain.data(), 1, coeff_modulus_size, getPowerOfTwo(coeff_count), ntt_tables);

        plain.parmsID() = parms_id;
    }

    void EvaluatorCuda::transformToNttInplace(CiphertextCuda &encrypted) const
    {
        encrypted.seed() = 0;

        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted is not valid for encryption parameters");
        }
        if (encrypted.isNttForm())
        {
            throw invalid_argument("encrypted is already in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_data_ptr;
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.size();

        auto ntt_tables = context_data.smallNTTTables();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        // Transform each polynomial to NTT domain
        kernel_util::kNttNegacyclicHarvey(encrypted.data(), encrypted_size, coeff_modulus_size, getPowerOfTwo(coeff_count), ntt_tables);

        // Finally change the is_ntt_transformed flag
        encrypted.isNttForm() = true;
    }

    void EvaluatorCuda::transformFromNttInplace(CiphertextCuda &encrypted_ntt) const
    {
        encrypted_ntt.seed() = 0;

        auto context_data_ptr = context_.getContextData(encrypted_ntt.parmsID());
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted_ntt is not valid for encryption parameters");
        }
        if (!encrypted_ntt.isNttForm())
        {
            throw invalid_argument("encrypted_ntt is not in NTT form");
        }

        // Extract encryption parameters.
        auto &context_data = *context_data_ptr;
        auto &parms = context_data.parms();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = parms.coeffModulus().size();
        size_t encrypted_ntt_size = encrypted_ntt.size();

        auto ntt_tables = context_data.smallNTTTables();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        // Transform each polynomial from NTT domain
        kernel_util::kInverseNttNegacyclicHarvey(encrypted_ntt.data(), encrypted_ntt_size, coeff_modulus_size, getPowerOfTwo(coeff_count), ntt_tables);

        // Finally change the is_ntt_transformed flag
        encrypted_ntt.isNttForm() = false;
    }
    

    void EvaluatorCuda::applyGaloisInplace(
        CiphertextCuda &encrypted, uint32_t galois_elt, const GaloisKeysCuda &galois_keys) const
    {
        encrypted.seed() = 0;

        // Don't validate all of galois_keys but just check the parms_id.
        if (galois_keys.parmsID() != context_.keyParmsID())
        {
            throw invalid_argument("galois_keys is not valid for encryption parameters");
        }

        auto &context_data = *context_.getContextData(encrypted.parmsID());
        auto &parms = context_data.parms();
        auto &coeff_modulus = parms.coeffModulus();
        size_t coeff_count = parms.polyModulusDegree();
        size_t coeff_modulus_size = coeff_modulus.size();
        size_t encrypted_size = encrypted.size();
        // Use key_context_data where permutation tables exist since previous runs.
        auto galois_tool = context_.keyContextData()->galoisTool();

        // Size check
        if (!productFitsIn(coeff_count, coeff_modulus_size))
        {
            throw logic_error("invalid parameters");
        }

        // Check if Galois key is generated or not.
        if (!galois_keys.hasKey(galois_elt))
        {
            throw invalid_argument("Galois key not present");
        }

        uint64_t m = mul_safe(static_cast<uint64_t>(coeff_count), uint64_t(2));

        // Verify parameters
        if (!(galois_elt & 1) || galois_elt >= m)
        {
            throw invalid_argument("Galois element is not valid");
        }
        if (encrypted_size > 2)
        {
            throw invalid_argument("encrypted size must be 2");
        }

        // SEAL_ALLOCATE_GET_RNS_ITER(temp, coeff_count, coeff_modulus_size, pool);
        temp_.ensure(coeff_modulus_size * coeff_count);
        auto temp = temp_.get();

        // DO NOT CHANGE EXECUTION ORDER OF FOLLOWING SECTION
        // BEGIN: Apply Galois for each ciphertext
        // Execution order is sensitive, since apply_galois is not inplace!
        if (parms.scheme() == SchemeType::bfv || parms.scheme() == SchemeType::bgv)
        {
            // !!! DO NOT CHANGE EXECUTION ORDER!!!

            // First transform encrypted.data(0)
            // auto encrypted_iter = iter(encrypted);
            galois_tool->applyGalois(encrypted.data(0), 1, coeff_modulus_size, galois_elt, coeff_modulus.asPointer(), temp);

            // Copy result to encrypted.data(0)
            kernel_util::kSetPolyArray(temp, 1, coeff_count, coeff_modulus_size, encrypted.data(0));

            // Next transform encrypted.data(1)
            galois_tool->applyGalois(encrypted.data(1), 1, coeff_modulus_size, galois_elt, coeff_modulus.asPointer(), temp);
        }
        else if (parms.scheme() == SchemeType::ckks)
        {
            // !!! DO NOT CHANGE EXECUTION ORDER!!!

            // First transform encrypted.data(0)
            // auto encrypted_iter = iter(encrypted);
            galois_tool->applyGaloisNtt(encrypted.data(0), 1, coeff_modulus_size, galois_elt, temp);

            // Copy result to encrypted.data(0)
            kernel_util::kSetPolyArray(temp, 1, coeff_count, coeff_modulus_size, encrypted.data(0));

            // Next transform encrypted.data(1)
            galois_tool->applyGaloisNtt(encrypted.data(1), 1, coeff_modulus_size, galois_elt, temp);
        }
        else
        {
            throw logic_error("scheme not implemented");
        }

        // Wipe encrypted.data(1)
        kernel_util::kSetZeroPolyArray(1, coeff_modulus_size, coeff_count, encrypted.data(1));

        // END: Apply Galois for each ciphertext
        // REORDERING IS SAFE NOW

        // Calculate (temp * galois_key[0], temp * galois_key[1]) + (ct[0], 0)
        switchKeyInplace(
            encrypted, temp, static_cast<const KSwitchKeysCuda &>(galois_keys), GaloisKeys::getIndex(galois_elt));
    }

    void EvaluatorCuda::rotateInternal(
        CiphertextCuda &encrypted, int steps, const GaloisKeysCuda &galois_keys) const
    {
        encrypted.seed() = 0;
        auto context_data_ptr = context_.getContextData(encrypted.parmsID());
        if (!context_data_ptr)
        {
            throw invalid_argument("encrypted is not valid for encryption parameters");
        }
        if (!context_data_ptr->qualifiers().using_batching)
        {
            throw logic_error("encryption parameters do not support batching");
        }
        if (galois_keys.parmsID() != context_.keyParmsID())
        {
            throw invalid_argument("galois_keys is not valid for encryption parameters");
        }

        // Is there anything to do?
        if (steps == 0)
        {
            return;
        }

        size_t coeff_count = context_data_ptr->parms().polyModulusDegree();
        auto galois_tool = context_data_ptr->galoisTool();

        // Check if Galois key is generated or not.
        if (galois_keys.hasKey(galois_tool->getEltFromStep(steps)))
        {
            // Perform rotation and key switching
            applyGaloisInplace(encrypted, galois_tool->getEltFromStep(steps), galois_keys);
        }
        else
        {
            // Convert the steps to NAF: guarantees using smallest HW
            std::vector<int> naf_steps = naf(steps);

            // If naf_steps contains only one element, then this is a power-of-two
            // rotation and we would have expected not to get to this part of the
            // if-statement.
            if (naf_steps.size() == 1)
            {
                throw invalid_argument("Galois key not present");
            }

            for (size_t i = 0; i < naf_steps.size(); i++) {
            // SEAL_ITERATE(naf_steps.cbegin(), naf_steps.size(), [&](auto step) {
                // We might have a NAF-term of size coeff_count / 2; this corresponds
                // to no rotation so we skip it. Otherwise call rotate_internal.
                if (safe_cast<size_t>(abs(naf_steps[i])) != (coeff_count >> 1))
                {
                    // Apply rotation for this step
                    this->rotateInternal(encrypted, naf_steps[i], galois_keys);
                }
            }
        }
    }

}