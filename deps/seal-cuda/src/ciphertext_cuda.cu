#include "ciphertext_cuda.cuh"
#include "serialize.h"
#include "utils/rlwe_cuda.cuh"
#include "evaluator_cuda.cuh"

namespace troy {

    void CiphertextCuda::save(std::ostream& stream) const {
        savet(stream, &parms_id_);
        savet(stream, &is_ntt_form_);
        savet(stream, &size_);
        savet(stream, &poly_modulus_degree_);
        savet(stream, &coeff_modulus_size_);
        savet(stream, &scale_);
        savet(stream, &correction_factor_);
        savet(stream, &seed_);
        bool terms = false;
        savet(stream, &terms);
        if (seed_ != 0 && size_ > 2) {
            throw std::invalid_argument("Seed exists but size is not 2.");
        }
        if (seed_ != 0) {
            util::HostArray<uint64_t> r(poly_modulus_degree_ * coeff_modulus_size_);
            KernelProvider::retrieve(r.get(), data_.get(), r.size());
            size_t dataSize = r.size();
            savet(stream, &dataSize);
            stream.write(reinterpret_cast<char*>(r.get()), sizeof(CiphertextCuda::ct_coeff_type) * r.size());
        } else {
            auto r = data_.toHost();
            size_t dataSize = r.size();
            savet(stream, &dataSize);
            stream.write(reinterpret_cast<char*>(r.begin()), sizeof(CiphertextCuda::ct_coeff_type) * r.size());
        }
    }

    void CiphertextCuda::saveTerms(std::ostream& stream, EvaluatorCuda& evaluator, const std::vector<size_t>& termIds) const {
        savet(stream, &parms_id_);
        savet(stream, &is_ntt_form_);

        util::HostDynamicArray<ct_coeff_type> r;

        if (is_ntt_form_) {
            CiphertextCuda copy = *this;
            evaluator.transformFromNttInplace(copy);
            r = copy.data_.toHost();
        } else {
            r = data_.toHost();
        }

        savet(stream, &size_);
        savet(stream, &poly_modulus_degree_);
        savet(stream, &coeff_modulus_size_);
        savet(stream, &scale_);
        savet(stream, &correction_factor_);
        savet(stream, &seed_);
        bool terms = true;
        savet(stream, &terms);
        if (seed_ != 0) {
            throw std::invalid_argument("Seed is not zero.");
        }
        // save degree 0 terms
        for (size_t id: termIds) {
            for (size_t j = 0; j < coeff_modulus_size_; j++) {
                auto num = r[j * poly_modulus_degree_ + id];
                stream.write(reinterpret_cast<char*>(&num), sizeof(decltype(num)));
            }
        }
        size_t offset = poly_modulus_degree_ * coeff_modulus_size_;
        size_t dataSize = r.size() - offset;
        savet(stream, &dataSize);
        stream.write(reinterpret_cast<char*>(r.begin() + offset), sizeof(CiphertextCuda::ct_coeff_type) * dataSize);
    }

    void CiphertextCuda::load(std::istream& stream) {
        seed_ = 0;
        loadt(stream, &parms_id_);
        loadt(stream, &is_ntt_form_);
        loadt(stream, &size_);
        loadt(stream, &poly_modulus_degree_);
        loadt(stream, &coeff_modulus_size_);
        loadt(stream, &scale_);
        loadt(stream, &correction_factor_);
        uint64_t seed; loadt(stream, &seed);
        bool terms; loadt(stream, &terms);
        if (terms) throw std::invalid_argument("Trying to load a termed ciphertext, but indices is not specified");
        if (seed == 0) {
            size_t dataSize;
            loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), dataSize);
        } else {
            throw std::invalid_argument("seed is not zero.");
        }
    }

    void CiphertextCuda::loadTerms(std::istream& stream, EvaluatorCuda& evaluator, const std::vector<size_t>& termIds) {
        seed_ = 0;
        loadt(stream, &parms_id_);
        loadt(stream, &is_ntt_form_);
        loadt(stream, &size_);
        loadt(stream, &poly_modulus_degree_);
        loadt(stream, &coeff_modulus_size_);
        loadt(stream, &scale_);
        loadt(stream, &correction_factor_);
        uint64_t seed; loadt(stream, &seed);
        bool terms; loadt(stream, &terms);
        if (!terms) throw std::invalid_argument("Trying to load a normal ciphertext, but term indices is specified");

        util::HostArray<ct_coeff_type> host(coeff_modulus_size_ * poly_modulus_degree_ * size_);
        // load degree 0 terms
        for (size_t id: termIds) {
            for (size_t j = 0; j < coeff_modulus_size_; j++) {
                ct_coeff_type num;
                stream.read(reinterpret_cast<char*>(&num), sizeof(decltype(num)));
                host[j * poly_modulus_degree_ + id] = num;
            }
        }
        // load terms degree greater than 0
        if (seed == 0) {
            size_t offset = poly_modulus_degree_ * coeff_modulus_size_;
            size_t dataSize;
            loadt(stream, &dataSize);
            stream.read(reinterpret_cast<char*>(host.get() + offset), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), host.size());
        } else {
            throw std::invalid_argument("seed is not zero.");
        }

        if (is_ntt_form_) {
            is_ntt_form_ = false;
            evaluator.transformToNttInplace(*this);
        }
    }

    void CiphertextCuda::load(std::istream& stream, const SEALContextCuda& context) {
        seed_ = 0;
        loadt(stream, &parms_id_);
        loadt(stream, &is_ntt_form_);
        loadt(stream, &size_);
        loadt(stream, &poly_modulus_degree_);
        loadt(stream, &coeff_modulus_size_);
        loadt(stream, &scale_);
        loadt(stream, &correction_factor_);
        uint64_t seed; loadt(stream, &seed);
        bool terms; loadt(stream, &terms);
        if (terms) throw std::invalid_argument("Trying to load a termed ciphertext, but indices is not specified");
        if (seed == 0) {
            size_t dataSize;
            loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), dataSize);
        } else {
            if (size_ > 2) throw std::invalid_argument("Seed exists but size is not 2.");
            size_t dataSize;
            loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(2 * poly_modulus_degree_ * coeff_modulus_size_);
            KernelProvider::copy(data_.get(), host.get(), dataSize);
            util::DeviceArray<hiprandState> curandStates(poly_modulus_degree_ * coeff_modulus_size_);
            auto& modulus = context.getContextData(parms_id_)->parms().coeffModulus();
            util::sampler::setupCurandStates(curandStates.get(), poly_modulus_degree_, seed);
            util::sampler::kSamplePolyUniform(curandStates.get(), modulus.size(), poly_modulus_degree_, modulus, data(1));

            auto &context_data = *context.getContextData(parms_id_);
            auto &parms = context_data.parms();
            auto &coeff_modulus = parms.coeffModulus();
            auto &plain_modulus = parms.plainModulus();
            size_t coeff_modulus_size = coeff_modulus.size();
            size_t coeff_count = parms.polyModulusDegree();
            size_t coeff_power = util::getPowerOfTwo(coeff_count);
            auto ntt_tables = context_data.smallNTTTables();
            SchemeType type = parms.scheme();

            if (type == SchemeType::bfv) {
                kernel_util::kInverseNttNegacyclicHarvey(data(1), 1, coeff_modulus_size, coeff_power, ntt_tables);
            }
        }
    }

}