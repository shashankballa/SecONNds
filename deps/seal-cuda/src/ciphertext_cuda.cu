#include "ciphertext_cuda.cuh"
#include "serialize.h"
#include "utils/rlwe_cuda.cuh"
#include "evaluator_cuda.cuh"

namespace troy {

    template <typename T>
    inline void _savet(std::ostream& stream, const T* obj) {
        stream.write(reinterpret_cast<const char*>(obj), sizeof(T));
    }
    
    template <typename T>
    inline void _loadt(std::istream& stream, T* obj) {
        stream.read(reinterpret_cast<char*>(obj), sizeof(T));
    }

    void CiphertextCuda::save(std::ostream& stream) const {
        _savet(stream, &parms_id_);
        _savet(stream, &is_ntt_form_);
        _savet(stream, &size_);
        _savet(stream, &poly_modulus_degree_);
        _savet(stream, &coeff_modulus_size_);
        _savet(stream, &scale_);
        _savet(stream, &correction_factor_);
        _savet(stream, &seed_);
        bool terms = false;
        _savet(stream, &terms);
        if (seed_ != 0 && size_ > 2) {
            throw std::invalid_argument("Seed exists but size is not 2.");
        }
        if (seed_ != 0) {
            util::HostArray<uint64_t> r(poly_modulus_degree_ * coeff_modulus_size_);
            KernelProvider::retrieve(r.get(), data_.get(), r.size());
            size_t dataSize = r.size();
            _savet(stream, &dataSize);
            stream.write(reinterpret_cast<char*>(r.get()), sizeof(CiphertextCuda::ct_coeff_type) * r.size());
        } else {
            auto r = data_.toHost();
            size_t dataSize = r.size();
            _savet(stream, &dataSize);
            stream.write(reinterpret_cast<char*>(r.begin()), sizeof(CiphertextCuda::ct_coeff_type) * r.size());
        }
    }

    void CiphertextCuda::load(std::istream& stream) {
        seed_ = 0;
        _loadt(stream, &parms_id_);
        _loadt(stream, &is_ntt_form_);
        _loadt(stream, &size_);
        _loadt(stream, &poly_modulus_degree_);
        _loadt(stream, &coeff_modulus_size_);
        _loadt(stream, &scale_);
        _loadt(stream, &correction_factor_);
        uint64_t seed; _loadt(stream, &seed);
        bool terms; _loadt(stream, &terms);
        if (terms) throw std::invalid_argument("Trying to load a termed ciphertext, but indices is not specified");
        if (seed == 0) {
            size_t dataSize;
            _loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), dataSize);
        } else {
            throw std::invalid_argument("seed is not zero.");
        }
    }

    void CiphertextCuda::saveTerms(std::ostream& stream, EvaluatorCuda& evaluator, const std::vector<size_t>& termIds) const {
        _savet(stream, &parms_id_);
        _savet(stream, &is_ntt_form_);

        util::HostDynamicArray<ct_coeff_type> r;

        if (is_ntt_form_) {
            CiphertextCuda copy = *this;
            evaluator.transformFromNttInplace(copy);
            r = copy.data_.toHost();
        } else {
            r = data_.toHost();
        }

        _savet(stream, &size_);
        _savet(stream, &poly_modulus_degree_);
        _savet(stream, &coeff_modulus_size_);
        _savet(stream, &scale_);
        _savet(stream, &correction_factor_);
        _savet(stream, &seed_);
        bool terms = true;
        _savet(stream, &terms);
        if (seed_ != 0) {
            throw std::invalid_argument("Seed is not zero.");
        }
        // save degree 0 terms
        for (size_t id: termIds) {
            for (size_t j = 0; j < coeff_modulus_size_; j++) {
                auto num = r[j * poly_modulus_degree_ + id];
                stream.write(reinterpret_cast<char*>(&num), sizeof(decltype(num)));
            }
        }
        size_t offset = poly_modulus_degree_ * coeff_modulus_size_;
        size_t dataSize = r.size() - offset;
        _savet(stream, &dataSize);
        stream.write(reinterpret_cast<char*>(r.begin() + offset), sizeof(CiphertextCuda::ct_coeff_type) * dataSize);
    }

    void CiphertextCuda::loadTerms(std::istream& stream, EvaluatorCuda& evaluator, const std::vector<size_t>& termIds) {
        seed_ = 0;
        _loadt(stream, &parms_id_);
        _loadt(stream, &is_ntt_form_);
        _loadt(stream, &size_);
        _loadt(stream, &poly_modulus_degree_);
        _loadt(stream, &coeff_modulus_size_);
        _loadt(stream, &scale_);
        _loadt(stream, &correction_factor_);
        uint64_t seed; _loadt(stream, &seed);
        bool terms; _loadt(stream, &terms);
        if (!terms) throw std::invalid_argument("Trying to load a normal ciphertext, but term indices is specified");

        util::HostArray<ct_coeff_type> host(coeff_modulus_size_ * poly_modulus_degree_ * size_);
        // load degree 0 terms
        for (size_t id: termIds) {
            for (size_t j = 0; j < coeff_modulus_size_; j++) {
                ct_coeff_type num;
                stream.read(reinterpret_cast<char*>(&num), sizeof(decltype(num)));
                host[j * poly_modulus_degree_ + id] = num;
            }
        }
        // load terms degree greater than 0
        if (seed == 0) {
            size_t offset = poly_modulus_degree_ * coeff_modulus_size_;
            size_t dataSize;
            _loadt(stream, &dataSize);
            stream.read(reinterpret_cast<char*>(host.get() + offset), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), host.size());
        } else {
            throw std::invalid_argument("seed is not zero.");
        }

        if (is_ntt_form_) {
            is_ntt_form_ = false;
            evaluator.transformToNttInplace(*this);
        }
    }

    void CiphertextCuda::load(std::istream& stream, const SEALContextCuda& context) {
        seed_ = 0;
        _loadt(stream, &parms_id_);
        _loadt(stream, &is_ntt_form_);
        _loadt(stream, &size_);
        _loadt(stream, &poly_modulus_degree_);
        _loadt(stream, &coeff_modulus_size_);
        _loadt(stream, &scale_);
        _loadt(stream, &correction_factor_);
        uint64_t seed; _loadt(stream, &seed);
        bool terms; _loadt(stream, &terms);
        if (terms) throw std::invalid_argument("Trying to load a termed ciphertext, but indices is not specified");
        if (seed == 0) {
            size_t dataSize;
            _loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(host.size());
            KernelProvider::copy(data_.get(), host.get(), dataSize);
        } else {
            if (size_ > 2) throw std::invalid_argument("Seed exists but size is not 2.");
            size_t dataSize;
            _loadt(stream, &dataSize);
            util::HostArray<ct_coeff_type> host(dataSize);
            stream.read(reinterpret_cast<char*>(host.get()), dataSize * sizeof(ct_coeff_type));
            data_.ensure(2 * poly_modulus_degree_ * coeff_modulus_size_);
            KernelProvider::copy(data_.get(), host.get(), dataSize);
            util::DeviceArray<hiprandState> curandStates(poly_modulus_degree_ * coeff_modulus_size_);
            auto& modulus = context.getContextData(parms_id_)->parms().coeffModulus();
            util::sampler::setupCurandStates(curandStates.get(), poly_modulus_degree_, seed);
            util::sampler::kSamplePolyUniform(curandStates.get(), modulus.size(), poly_modulus_degree_, modulus, data(1));

            auto &context_data = *context.getContextData(parms_id_);
            auto &parms = context_data.parms();
            auto &coeff_modulus = parms.coeffModulus();
            auto &plain_modulus = parms.plainModulus();
            size_t coeff_modulus_size = coeff_modulus.size();
            size_t coeff_count = parms.polyModulusDegree();
            size_t coeff_power = util::getPowerOfTwo(coeff_count);
            auto ntt_tables = context_data.smallNTTTables();
            SchemeType type = parms.scheme();

            if (type == SchemeType::bfv) {
                kernel_util::kInverseNttNegacyclicHarvey(data(1), 1, coeff_modulus_size, coeff_power, ntt_tables);
            }
        }
    }
}