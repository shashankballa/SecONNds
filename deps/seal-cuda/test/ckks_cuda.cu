#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT license.

#include "../src/troy_cuda.cuh"
#include <ctime>
#include <vector>
#include "gtest/gtest.h"

using namespace troyn;
using namespace std;

namespace troytest
{
    TEST(CKKSEncoderCudaTest, CKKSEncoderEncodeVectorDecodeTest)
    {
        KernelProvider::initialize();
        EncryptionParameters parms(SchemeType::ckks);
        {
            size_t slots = 32;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 40, 40, 40, 40 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(0.0, 0.0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 16);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            size_t slots = 32;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 60, 60, 60, 60 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 40);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            size_t slots = 64;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 60, 60, 60 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 40);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            size_t slots = 64;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 30, 30, 30, 30, 30 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 40);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            size_t slots = 32;
            parms.setPolyModulusDegree(128);
            parms.setCoeffModulus(CoeffModulus::Create(128, { 30, 30, 30, 30, 30 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 40);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            // Many primes
            size_t slots = 32;
            parms.setPolyModulusDegree(128);
            parms.setCoeffModulus(CoeffModulus::Create(
                128, { 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30, 30 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            double delta = (1ULL << 40);
            Plaintext plain;
            encoder.encode(values, context.firstParmsID(), delta, plain);
            vector<complex<double>> result;
            encoder.decode(plain, result);

            for (size_t i = 0; i < slots; ++i)
            {
                auto tmp = abs(values[i].real() - result[i].real());
                ASSERT_TRUE(tmp < 0.5);
            }
        }
        {
            size_t slots = 64;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 40, 40, 40, 40, 40 }));
            SEALContext context(parms, false, SecurityLevel::none);

            vector<complex<double>> values(slots);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 20);

            for (size_t i = 0; i < slots; i++)
            {
                complex<double> value(static_cast<double>(rand() % data_bound), 0);
                values[i] = value;
            }

            CKKSEncoder encoder(context);
            {
                // Use a very large scale
                double delta = pow(2.0, 110);
                Plaintext plain;
                encoder.encode(values, context.firstParmsID(), delta, plain);
                vector<complex<double>> result;
                encoder.decode(plain, result);

                for (size_t i = 0; i < slots; ++i)
                {
                    auto tmp = abs(values[i].real() - result[i].real());
                    ASSERT_TRUE(tmp < 0.5);
                }
            }
            {
                // Use a scale over 128 bits
                double delta = pow(2.0, 130);
                Plaintext plain;
                encoder.encode(values, context.firstParmsID(), delta, plain);
                vector<complex<double>> result;
                encoder.decode(plain, result);

                for (size_t i = 0; i < slots; ++i)
                {
                    auto tmp = abs(values[i].real() - result[i].real());
                    ASSERT_TRUE(tmp < 0.5);
                }
            }
        }
    }

    TEST(CKKSEncoderCudaTest, CKKSEncoderEncodeSingleDecodeTest)
    {
        KernelProvider::initialize();
        EncryptionParameters parms(SchemeType::ckks);
        {
            size_t slots = 16;
            parms.setPolyModulusDegree(64);
            parms.setCoeffModulus(CoeffModulus::Create(64, { 40, 40, 40, 40 }));
            SEALContext context(parms, false, SecurityLevel::none);
            CKKSEncoder encoder(context);

            srand(static_cast<unsigned>(time(NULL)));
            int data_bound = (1 << 30);
            double delta = (1ULL << 16);
            Plaintext plain;
            vector<complex<double>> result;

            for (int iRun = 0; iRun < 50; iRun++)
            {
                double value = static_cast<double>(rand() % data_bound);
                encoder.encode(value, context.firstParmsID(), delta, plain);
                encoder.decode(plain, result);

                for (size_t i = 0; i < slots; ++i)
                {
                    auto tmp = abs(value - result[i].real());
                    ASSERT_TRUE(tmp < 0.5);
                }
            }
        }
        {
            size_t slots = 32;
            parms.setPolyModulusDegree(slots << 1);
            parms.setCoeffModulus(CoeffModulus::Create(slots << 1, { 40, 40, 40, 40 }));
            SEALContext context(parms, false, SecurityLevel::none);
            CKKSEncoder encoder(context);

            srand(static_cast<unsigned>(time(NULL)));
            {
                int data_bound = (1 << 30);
                Plaintext plain;
                vector<complex<double>> result;

                for (int iRun = 0; iRun < 50; iRun++)
                {
                    int value = static_cast<int>(rand() % data_bound);
                    encoder.encode(value, context.firstParmsID(), plain);
                    encoder.decode(plain, result);

                    for (size_t i = 0; i < slots; ++i)
                    {
                        auto tmp = abs(value - result[i].real());
                        ASSERT_TRUE(tmp < 0.5);
                    }
                }
            }
            {
                // Use a very large scale
                int data_bound = (1 << 20);
                Plaintext plain;
                vector<complex<double>> result;

                for (int iRun = 0; iRun < 50; iRun++)
                {
                    int value = static_cast<int>(rand() % data_bound);
                    encoder.encode(value, context.firstParmsID(), plain);
                    encoder.decode(plain, result);

                    for (size_t i = 0; i < slots; ++i)
                    {
                        auto tmp = abs(value - result[i].real());
                        ASSERT_TRUE(tmp < 0.5);
                    }
                }
            }
            {
                // Use a scale over 128 bits
                int data_bound = (1 << 20);
                Plaintext plain;
                vector<complex<double>> result;

                for (int iRun = 0; iRun < 50; iRun++)
                {
                    int value = static_cast<int>(rand() % data_bound);
                    encoder.encode(value, context.firstParmsID(), plain);
                    encoder.decode(plain, result);

                    for (size_t i = 0; i < slots; ++i)
                    {
                        auto tmp = abs(value - result[i].real());
                        ASSERT_TRUE(tmp < 0.5);
                    }
                }
            }
        }
    }
} // namespace sealtest
