#include "hip/hip_runtime.h"
#include "../src/troy_cuda.cuh"
#include <vector>
#include <string>
#include <sys/time.h>
#include <cassert>
#include <map>
#include <complex>
#include <iomanip>

using namespace troyn;
using std::vector;
using std::complex;

namespace troytest {
    
    template<typename T> 
    vector<T> vectorAdd(const vector<T> a, const vector<T> b) {
        assert(a.size() == b.size());
        vector<T> ret; ret.reserve(a.size());
        for (size_t i = 0; i < a.size(); i++) ret.push_back(a[i] + b[i]);
        return ret;
    }

    template<typename T> 
    vector<T> vectorMultiply(const vector<T> a, const vector<T> b) {
        assert(a.size() == b.size());
        vector<T> ret; ret.reserve(a.size());
        for (size_t i = 0; i < a.size(); i++) ret.push_back(a[i] * b[i]);
        return ret;
    }

    template<typename T> 
    vector<T> vectorSub(const vector<T> a, const vector<T> b) {
        assert(a.size() == b.size());
        vector<T> ret; ret.reserve(a.size());
        for (size_t i = 0; i < a.size(); i++) ret.push_back(a[i] - b[i]);
        return ret;
    }

    template<typename T> 
    vector<T> vectorNegate(const vector<T> a) {
        vector<T> ret; ret.reserve(a.size());
        for (size_t i = 0; i < a.size(); i++) ret.push_back(-a[i]);
        return ret;
    }

    template<typename T>
    vector<T> vectorRotate(const vector<T>& a, size_t n) {
        vector<T> ret; ret.reserve(a.size());
        ret.insert(ret.end(), a.begin() + n, a.end());
        ret.insert(ret.end(), a.begin(), a.begin() + n);
    }

    template<typename T> 
    inline bool is_equal(const vector<T>& a, const vector<T>& b, double eps = 1e-6) {
        if (a.size() != b.size()) return false;
        for (size_t i = 0; i < a.size(); i++) {
            if (std::abs(a[i] - b[i]) > eps) return false;
        }
        return true;
    }

    template<typename T> 
    inline bool is_zero(const vector<T>& a, double eps = 1e-6) {
        for (size_t i = 0; i < a.size(); i++) {
            if (std::abs(a[i]) > eps) return false;
        }
        return true;
    }

    template<typename T>
    void printVector(const vector<T>& r, bool full = false) {
        std::cout << "[";
        for (size_t i = 0; i < r.size(); i++) {
            if (r.size() > 8 && !full && i == 4) {
                std::cout << " ...";
                i = r.size() - 4;
            }
            if (i!=0) std::cout << ", ";
            std::cout << std::setprecision(3) << std::fixed << (double) r[i];
        }
        std::cout << "]" << std::endl;
    }

    inline std::string pass_str(bool pass) {
        std::stringstream ss;
        // color red if failed, green if passed
        ss << (pass ? "\033[1;32m" : "\033[1;31m") << (pass ? "PASS" : "FAIL") << "\033[0m";
        return ss.str();
    }

    class Timer {
    public:
        std::vector<timeval> times;
        std::vector<double> accumulated; // ms
        std::vector<std::string> names;
        Timer() {}
        long registerTimer(std::string name = "") {
            times.push_back(timeval()); 
            accumulated.push_back(0);
            int ret = times.size() - 1;
            names.push_back(name);
            return ret;
        }
        void tick(long i = 0) {
            if (times.size() < 1) registerTimer();
            assert(i < times.size());
            gettimeofday(&times[i], 0);
        }
        double tock(long i = 0) {
            assert(i < times.size());
            timeval s; gettimeofday(&s, 0);
            auto timeElapsed = (s.tv_sec - times[i].tv_sec) * 1000.0;
            timeElapsed += (s.tv_usec - times[i].tv_usec) / 1000.0;
            accumulated[i] += timeElapsed;
            return accumulated[i];
        }
        
        void clear() {
            times.clear();
            accumulated.clear();
            names.clear();
        }

        std::map<std::string, double> gather(double divisor = 1) {
            std::map<std::string, double> p;
            for (long i=0; i<times.size(); i++) {
                p[names[i]] = accumulated[i] / divisor;
            }
            clear();
            return p;
        }
    };

    class TimeTest {
        
    protected:
        Timer tim;
        Encryptor* encryptor;
        Decryptor* decryptor;
        Evaluator* evaluator;
        SEALContext* context;
        RelinKeys rlk;
        PublicKey pk;
        GaloisKeys gk;
        KeyGenerator* keygen;

    public:
        TimeTest() {
            tim.clear();
            encryptor = nullptr;
            evaluator = nullptr;
            context = nullptr;
            decryptor = nullptr;
        }

        ~TimeTest() {
            if (encryptor) delete encryptor;
            if (evaluator) delete evaluator;
            if (context) delete context;
            if (decryptor) delete decryptor;
            if (keygen) delete keygen;
        }

        virtual Plaintext randomPlaintext() = 0;
        virtual Ciphertext randomCiphertext() = 0;
        // virtual void testEncode() = 0;

        void testEncrypt(int repeatCount = 1000) {
            auto p1 = randomPlaintext();
            Ciphertext c2;
            Plaintext p2;
            auto t1 = tim.registerTimer("Encrypt");
            auto t2 = tim.registerTimer("Decrypt");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                encryptor->encrypt(p1, c2);
                tim.tock(t1);
                tim.tick(t2);
                decryptor->decrypt(c2, p2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void printTimer(std::map<std::string, double> r) {
            for (auto& p: r) {
                std::cout << std::setw(25) << std::right << p.first << ":";
                std::cout << std::setw(10) << std::right << std::fixed << std::setprecision(3)
                    << p.second << std::endl;
            }
        }

        void testAdd(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto c2 = randomCiphertext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("Add-assign");
            auto t2 = tim.registerTimer("Add-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->add(c1, c2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->addInplace(c3, c1);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testAddPlain(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto p2 = randomPlaintext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("AddPlain-assign");
            auto t2 = tim.registerTimer("AddPlain-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->addPlain(c1, p2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->addPlainInplace(c3, p2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testMultiplyPlain(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto p2 = randomPlaintext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("MultiplyPlain-assign");
            auto t2 = tim.registerTimer("MultiplyPlain-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->multiplyPlain(c1, p2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->multiplyPlainInplace(c3, p2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testSquare(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            Ciphertext c2;
            Ciphertext c3;
            auto t1 = tim.registerTimer("Square-assign");
            auto t2 = tim.registerTimer("Square-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->square(c1, c2);
                tim.tock(t1);
                c3 = c1;
                tim.tick(t2);
                evaluator->squareInplace(c3);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testMemoryPool(int repeatCount = 1000) {
            auto t1 = tim.registerTimer("Preallocate");
            auto t2 = tim.registerTimer("Allocate");
            tim.tick(t1);
            auto c1 = randomCiphertext();
            Ciphertext c2;
            for (int t = 0; t < repeatCount; t++) {
                evaluator->square(c1, c2);
            }
            tim.tock(t1);
            tim.tick(t2);
            for (int t = 0; t < repeatCount; t++) {
                Ciphertext c3;
                evaluator->square(c1, c3);
            }
            tim.tock(t2);
            printTimer(tim.gather(repeatCount));
        }

    };

    class TimeTestCKKS: public TimeTest {

        CKKSEncoder* encoder;
        size_t slotCount;
        int dataBound;
        double delta;
    
    public:

        TimeTestCKKS(size_t polyModulusDegree, vector<int> qs, int dataBound = 1<<6, double delta=static_cast<double>(1<<16)) {
            KernelProvider::initialize();
            slotCount = polyModulusDegree / 2;
            this->dataBound = dataBound;
            this->delta = delta;
            EncryptionParameters parms(SchemeType::ckks);
            parms.setPolyModulusDegree(polyModulusDegree);
            parms.setCoeffModulus(CoeffModulus::Create(polyModulusDegree, qs));
            context = new SEALContext(parms);
            keygen = new KeyGenerator(*context);
            keygen->createPublicKey(pk);
            keygen->createRelinKeys(rlk);
            keygen->createGaloisKeys(gk);
            encoder = new CKKSEncoder(*context);
            encryptor = new Encryptor(*context, pk);
            decryptor = new Decryptor(*context, keygen->secretKey());
            evaluator = new Evaluator(*context);
        }

        ~TimeTestCKKS() {
            if (encoder) delete encoder;
        }
        
        static vector<complex<double>> randomVector(size_t count, int data_bound) {
            vector<complex<double>> input(count, 0.0);
            for (size_t i = 0; i < count; i++)
            {
                input[i] = static_cast<double>(rand() % data_bound);
            }
            return input;
        }

        Plaintext randomPlaintext() override {
            auto p = randomVector(slotCount, dataBound);
            Plaintext ret; encoder->encode(p, delta, ret);
            return std::move(ret);
        }

        Ciphertext randomCiphertext() override {
            auto r = randomPlaintext();
            Ciphertext ret; encryptor->encrypt(r, ret);
            return std::move(ret);
        }

        void testEncode(int repeatCount = 1000) {
            auto m1 = randomVector(slotCount, dataBound);
            auto m2 = randomVector(slotCount, dataBound);
            Plaintext p1;
            auto t1 = tim.registerTimer("Encode");
            auto t2 = tim.registerTimer("Decode");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                encoder->encode(m1, delta, p1);
                tim.tock(t1);
                tim.tick(t2);
                encoder->decode(p1, m2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testMultiplyRescale(int repeatCount = 100) {
            auto c1 = randomCiphertext();
            auto c2 = randomCiphertext();
            Ciphertext c3, c4;
            Ciphertext c5;
            auto t1 = tim.registerTimer("Multiply-assign");
            auto t2 = tim.registerTimer("Rescale-assign");
            auto t3 = tim.registerTimer("Multiply-inplace");
            auto t4 = tim.registerTimer("Rescale-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->multiply(c1, c2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->rescaleToNext(c3, c4);
                tim.tock(t2);
                c5 = c1;
                tim.tick(t3);
                evaluator->multiplyInplace(c5, c2);
                tim.tock(t3);
                tim.tick(t4);
                evaluator->rescaleToNextInplace(c5);
                tim.tock(t4);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testRotateVector(int repeatCount = 100) {
            auto c1 = randomCiphertext();
            Ciphertext c2;
            auto t1 = tim.registerTimer("Rotate-assign");
            auto t2 = tim.registerTimer("Rotate-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->rotateVector(c1, 1, gk, c2);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->rotateVectorInplace(c1, 1, gk);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));
        }

        void testAll() {
            this->testEncode();
            this->testEncrypt();
            this->testAdd();
            this->testAddPlain();
            this->testMultiplyRescale();
            this->testMultiplyPlain();
            this->testSquare();
            this->testRotateVector();
            this->testMemoryPool();
        }

    };



    class TimeTestBFVBGV: public TimeTest {

        BatchEncoder* encoder;
        size_t slotCount;
        int dataBound;
        double delta;
    
    public:

        TimeTestBFVBGV(bool bgv, size_t polyModulusDegree, uint64_t plainModulusBitSize, vector<int> qs, int dataBound = 1<<6) {
            KernelProvider::initialize();
            slotCount = polyModulusDegree;
            this->dataBound = dataBound;
            this->delta = delta;
            EncryptionParameters parms(bgv ? SchemeType::bgv : SchemeType::bfv);
            parms.setPolyModulusDegree(polyModulusDegree);
            parms.setPlainModulus(PlainModulus::Batching(polyModulusDegree, plainModulusBitSize));
            // parms.setCoeffModulus(CoeffModulus::BFVDefault(polyModulusDegree));
            parms.setCoeffModulus(CoeffModulus::Create(polyModulusDegree, qs));
            context = new SEALContext(parms);
            keygen = new KeyGenerator(*context);
            keygen->createPublicKey(pk);
            keygen->createRelinKeys(rlk);
            keygen->createGaloisKeys(gk);
            encoder = new BatchEncoder(*context);
            encryptor = new Encryptor(*context, pk);
            decryptor = new Decryptor(*context, keygen->secretKey());
            evaluator = new Evaluator(*context);
        }

        ~TimeTestBFVBGV() {
            if (encoder) delete encoder;
        }
        
        static vector<int64_t> randomVector(size_t count, int data_bound) {
            vector<int64_t> input(count, 0);
            for (size_t i = 0; i < count; i++)
            {
                input[i] = rand() % data_bound;
            }
            return input;
        }

        Plaintext randomPlaintext() override {
            auto p = randomVector(slotCount, dataBound);
            Plaintext ret; encoder->encode(p, ret);
            return std::move(ret);
        }

        Ciphertext randomCiphertext() override {
            auto r = randomPlaintext();
            Ciphertext ret; encryptor->encrypt(r, ret);
            return std::move(ret);
        }

        vector<int64_t> decode(const Plaintext& p) {
            vector<int64_t> r;
            encoder->decode(p, r);
            return r;
        }

        vector<int64_t> decrypt(const Ciphertext& c) {
            Ciphertext c2 = c;
            if (c.isNttForm()) {
                evaluator->transformFromNttInplace(c2);
            }
            Plaintext p;
            decryptor->decrypt(c2, p);
            return decode(p);
        }

        void testSaveLoad(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto t1 = tim.registerTimer("Save Ciphertext");
            auto t2 = tim.registerTimer("Load Ciphertext");
            Ciphertext c2;
            for (int t = 0; t < repeatCount; t++) {
                std::stringstream ss;
                tim.tick(t1);
                c1.save(ss);
                tim.tock(t1);
                tim.tick(t2);
                c2.load(ss);
                tim.tock(t2);
            }
            auto p3 = randomPlaintext();
            auto t3 = tim.registerTimer("Save Plaintext");
            auto t4 = tim.registerTimer("Load Plaintext");
            Plaintext p4;
            for (int t = 0; t < repeatCount; t++) {
                std::stringstream ss;
                tim.tick(t3);
                p3.save(ss);
                tim.tock(t3);
                tim.tick(t4);
                p4.load(ss);
                tim.tock(t4);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3, m4;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decode(p3);
            m4 = decode(p4);
            bool pass = is_equal(m1, m2);
            std::cout << pass_str(pass) << " | Save/Load Ciphertext" << std::endl;
            pass = is_equal(m3, m4);
            std::cout << pass_str(pass) << " | Save/Load Plaintext" << std::endl;
        }

        void testEncode(int repeatCount = 1000) {
            auto m1 = randomVector(slotCount, dataBound);
            // print m1
            std::cout << "m1: size=" << m1.size() << ", data="; printVector(m1);
            vector<int64_t> m2;
            auto t1 = tim.registerTimer("Encode");
            auto t2 = tim.registerTimer("Decode");
            for (int t = 0; t < repeatCount; t++) {
                Plaintext p1;
                tim.tick(t1);
                encoder->encode(m1, p1);
                tim.tock(t1);
                tim.tick(t2);
                encoder->decode(p1, m2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            // print m2
            std::cout << "m2: size=" << m2.size() << ", data="; printVector(m2);

            bool pass = is_equal(m1, m2);
            std::cout << pass_str(pass) << " | Encode/Decode" << std::endl;
        }
  
        void testAdd(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto c2 = randomCiphertext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("Add-assign");
            auto t2 = tim.registerTimer("Add-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->add(c1, c2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->addInplace(c3, c1);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decrypt(c3);

            bool pass = is_equal(m3, vectorAdd(vectorAdd(m1, m2), m1));
            std::cout << pass_str(pass) << " | Add" << std::endl;
        }

        void testAddPlain(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto p2 = randomPlaintext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("AddPlain-assign");
            auto t2 = tim.registerTimer("AddPlain-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->addPlain(c1, p2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->addPlainInplace(c3, p2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decode(p2);
            m3 = decrypt(c3);

            bool pass = is_equal(m3, vectorAdd(vectorAdd(m1, m2), m2));
            std::cout << pass_str(pass) << " | AddPlain" << std::endl;
        }

        void testMultiplyPlain(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto p2 = randomPlaintext();
            Ciphertext c3;
            auto t1 = tim.registerTimer("MultiplyPlain-assign");
            auto t2 = tim.registerTimer("MultiplyPlain-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->multiplyPlain(c1, p2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->multiplyPlainInplace(c3, p2);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decode(p2);
            m3 = decrypt(c3);

            bool pass = is_equal(m3, vectorMultiply(vectorMultiply(m1, m2), m2));
            std::cout << pass_str(pass) << " | MultiplyPlain" << std::endl;
        }

        void testSquare(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            Ciphertext c2;
            Ciphertext c3;
            auto t1 = tim.registerTimer("Square-assign");
            auto t2 = tim.registerTimer("Square-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->square(c1, c2);
                tim.tock(t1);
                c3 = c1;
                tim.tick(t2);
                evaluator->squareInplace(c3);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decrypt(c3);

            auto m1_2 = vectorMultiply(m1, m1);
            bool pass = is_equal(m2, m1_2);
            std::cout << pass_str(pass) << " | Square" << std::endl;
            pass = is_equal(m3, m1_2);
            std::cout << pass_str(pass) << " | SquareInplace" << std::endl;
        }


        void testMultiplyRescale(int repeatCount = 100) {
            auto c1 = randomCiphertext();
            auto c2 = randomCiphertext();
            Ciphertext c3, c4;
            Ciphertext c5;
            auto t1 = tim.registerTimer("Multiply-assign");
            auto t2 = tim.registerTimer("ModSwitch-assign");
            auto t2_ = tim.registerTimer("Relinearize-assign");
            auto t3 = tim.registerTimer("Multiply-inplace");
            auto t4 = tim.registerTimer("ModSwitch-inplace");
            auto t4_ = tim.registerTimer("Relinearize-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->multiply(c1, c2, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->modSwitchToNext(c3, c4);
                tim.tock(t2);
                c5 = c1;
                tim.tick(t3);
                evaluator->multiplyInplace(c5, c2);
                tim.tock(t3);
                tim.tick(t4);
                evaluator->modSwitchToNextInplace(c5);
                tim.tock(t4);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3, m4, m5;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decrypt(c3);
            m4 = decrypt(c4);
            m5 = decrypt(c5);

            bool pass = is_equal(m3, vectorMultiply(m1, m2));
            std::cout << pass_str(pass) << " | Multiply" << std::endl;
            pass = is_equal(m4, m3);
            std::cout << pass_str(pass) << " | ModSwitch" << std::endl;
            pass = is_equal(m5, m3);
            std::cout << pass_str(pass) << " | Inplace" << std::endl;
        }

        void testRotateVector(int repeatCount = 100) {
            auto c1 = randomCiphertext();
            Ciphertext c2;
            auto t1 = tim.registerTimer("RotateRows-assign");
            auto t2 = tim.registerTimer("RotateRows-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                evaluator->rotateRows(c1, 1, gk, c2);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->rotateRowsInplace(c1, 1, gk);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            bool pass = is_equal(m1, m2);
            std::cout << pass_str(pass) << " | rotateRows" << std::endl;
        }
        
        /*
            Test the performance of transformToNtt and transformToNttInplace
            args:
                repeatCount: the number of times to repeat the test
        */
        void testToNtt(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            Ciphertext c2;
            Ciphertext c3;
            auto t1 = tim.registerTimer("ToNtt-assign");
            auto t2 = tim.registerTimer("ToNtt-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                // evaluator->square(c1, c2);
                evaluator->transformToNtt(c1, c2);
                tim.tock(t1);
                c3 = c1;
                tim.tick(t2);
                evaluator->transformToNttInplace(c3);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decrypt(c3);
            bool pass = is_equal(m2, m1);
            std::cout << pass_str(pass) << " | ToNTT" << std::endl;
            pass = is_equal(m3, m1);
            std::cout << pass_str(pass) << " | ToNTTInplace" << std::endl;
        }

        /*
            Test the performance of transformFromNtt and transformFromNttInplace
            args:
                repeatCount: the number of times to repeat the test
        */
        void testFromNtt(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            evaluator->transformToNttInplace(c1);
            Ciphertext c2;
            Ciphertext c3;
            auto t1 = tim.registerTimer("FromNTT-assign");
            auto t2 = tim.registerTimer("FromNTT-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t1);
                // evaluator->square(c1, c2);
                evaluator->transformFromNtt(c1, c2);
                tim.tock(t1);
                c3 = c1;
                tim.tick(t2);
                evaluator->transformFromNttInplace(c3);
                tim.tock(t2);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m3;
            m1 = decrypt(c1);
            m2 = decrypt(c2);
            m3 = decrypt(c3);
            bool pass = is_equal(m2, m1);
            std::cout << pass_str(pass) << " | FromNTT" << std::endl;
            pass = is_equal(m3, m1);
            std::cout << pass_str(pass) << " | FromNTTInplace" << std::endl;
        }

        void testMultiplyPlainNtt(int repeatCount = 1000) {
            auto c1 = randomCiphertext();
            auto p2 = randomPlaintext();
            Plaintext p2_ntt;
            Ciphertext c3, c4, c5;
            auto t0 = tim.registerTimer("ToNttPlain-assign");
            auto t1 = tim.registerTimer("ToNtt-assign");
            auto t2 = tim.registerTimer("MultiplyPlainNTT-assign");
            auto t3 = tim.registerTimer("FromNtt-assign");
            auto t4 = tim.registerTimer("ToNttPlain-inplace");
            auto t5 = tim.registerTimer("ToNtt-inplace");
            auto t6 = tim.registerTimer("MultiplyPlainNTT-inplace");
            auto t7 = tim.registerTimer("FromNtt-inplace");
            for (int t = 0; t < repeatCount; t++) {
                tim.tick(t0);
                evaluator->transformToNtt(p2, c1.parmsID(), p2_ntt);
                tim.tock(t0);
                tim.tick(t1);
                evaluator->transformToNtt(c1, c3);
                tim.tock(t1);
                tim.tick(t2);
                evaluator->multiplyPlain(c3, p2_ntt, c4);
                tim.tock(t2);
                tim.tick(t3);
                evaluator->transformFromNtt(c4, c5);
                tim.tock(t3);
                c5 = c1;
                p2_ntt = p2;
                tim.tick(t4);
                evaluator->transformToNttInplace(p2_ntt, c1.parmsID());
                tim.tock(t4);
                tim.tick(t5);
                evaluator->transformToNttInplace(c5);
                tim.tock(t5);
                tim.tick(t6);
                evaluator->multiplyPlainInplace(c5, p2_ntt);
                tim.tock(t6);
                tim.tick(t7);
                evaluator->transformFromNttInplace(c5);
                tim.tock(t7);
            }
            printTimer(tim.gather(repeatCount));

            vector<int64_t> m1, m2, m5;
            m1 = decrypt(c1);
            m2 = decode(p2);
            m5 = decrypt(c5);
            bool pass = is_equal(m5, vectorMultiply(m1, m2));
            std::cout << pass_str(pass) << " | MultiplyPlainNTT" << std::endl;
        }

        void testAll() {
            this->testSaveLoad();
            this->testEncode();
            this->testEncrypt();
            this->testAdd();
            this->testAddPlain();
            this->testMultiplyRescale();
            this->testMultiplyPlain();
            this->testSquare();
            this->testRotateVector();
            this->testToNtt();
            this->testFromNtt();
            this->testMultiplyPlainNtt();
            this->testMemoryPool();
        }
    };
}

int main() {

    std::vector<int> qs = {60, 60, 60};
    int plainModBits = 20;
    int polyModDeg = 8192;

    std::cout << "----- TimeTest cuda CKKS -----\n";
    troytest::TimeTestCKKS test(polyModDeg, qs);
    test.testAll();

    std::cout << "----- TimeTest cuda BFV -----\n";
    troytest::TimeTestBFVBGV test2(false, polyModDeg, plainModBits, qs);
    test2.testAll();

    std::cout << "----- TimeTest cuda BGV -----\n";
    troytest::TimeTestBFVBGV test3(true, polyModDeg, plainModBits, qs);
    test3.testAll();
    return 0;
}